#include "hip/hip_runtime.h"
/*
/*
Copyright 2008 Steven Brodhead, Jr.
Copyright 2011-2016 Steven Brodhead, Sr., Centcom Inc.

// All rights reserved.
 
//     Fractal Architect Render Engine - a GPU accelerated flame fractal renderer written in C++
//
//     This is free software; you can redistribute it and/or modify it under the terms of the GNU Lesser
//     General Public License as published by the Free Software Foundation; either version 2.1 of the
//     License, or (at your option) any later version.
//
//     This software is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY; without
//     even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the GNU
//     Lesser General Public License for more details.
//
//     You should have received a copy of the GNU Lesser General Public License along with this software;
//     if not, write to the Free Software Foundation, Inc., 51 Franklin St, Fifth Floor, Boston, MA
//     02110-1301 USA, or see the FSF site: http://www.fsf.org.
*/

/*
Extended version for the use from within JWildfire7+. Requires the extended FACLRenderJWF.exe as client to execute.
See https://bitbucket.org/amaschke/faengine/src/JWildfireExperiments/ for more details.
Copyright 2021 Andreas Maschke, with contributions made by Jesus Sosa.
*/

// the following switches are made to help to keep the kernel small and include only features which are actually used (="poor module system")
// activate noise features
//  #define ADD_FEATURE_CELLULAR_NOISE
//  #define ADD_FEATURE_CUBIC_NOISE
//  #define ADD_FEATURE_PERLIN_NOISE
//  #define ADD_FEATURE_SIMPLEX_NOISE
//  #define ADD_FEATURE_VALUE_NOISE
//  #define ADD_FEATURE_WHITE_NOISE
// activate wfields, please note that you must ensure to enable all the required noise types, otherwise all noise (and so the wfield) will be zero
//   #define ADD_FEATURE_WFIELDS
//   #define ADD_FEATURE_WFIELDS_JITTER
// activate additional features
//   #define ADD_FEATURE_DOF
// Usually, these switches are set by the client by replacing the following placeholder:
__GLOBAL_DEFINITIONS__

#if defined(ADD_FEATURE_CELLULAR_NOISE) || defined(ADD_FEATURE_CUBIC_NOISE) || defined(ADD_FEATURE_PERLIN_NOISE) || defined(ADD_FEATURE_SIMPLEX_NOISE) || defined(ADD_FEATURE_VALUE_NOISE) || defined(ADD_FEATURE_WHITE_NOISE)
  #define ADD_FEATURE_FAST_NOISE
#else
  #undef ADD_FEATURE_FAST_NOISE
#endif

#define NUM_ITERATIONS 100
// #define DENSITY_KERNAL_RADIUS 7
#define DENSITY_KERNAL_RADIUS_16KB 7
#define DENSITY_KERNAL_RADIUS_32KB 14
#define DENSITY_KERNAL_RADIUS_48KB 19
#define NUM_FRAMES 160
#define FRAME_RATE 30
#define BITRATE 54000000

#ifndef SUPERSAMPLE_WIDTH
#define SUPERSAMPLE_WIDTH 0.25f
#endif

#ifndef FLAMEDATA_H
#define FLAMEDATA_H

#define MAX_XFORMS 58 // We're limited to 64KB constant memory for compute capacity 1.0.
// All xForms must fit in this.


#define NO_RGBA_CONSTRUCTOR

#define uint  unsigned
#define ulong unsigned long

#define M_PI_F    3.141592653589793f
#define M_PI_2_F  1.5707963267949f
#define M_PI_4_F  0.78539816339745f
#define M_1_PI_F  0.31830988618379f
#define M_2_PI_F  0.63661977236758f

#define JWF_EXTENSIONS

#ifndef RGBA_H
#define RGBA_H

struct  __align__(16) rgba
{
    float r;
    float g;
    float b;
    float a;
};

#endif

#ifdef JWF_EXTENSIONS
__device__ float sqrtf_safe(float x) {
  if (x <= 0.0f)
    return 0.0f;
  else	
    return sqrtf(x);
}

__device__ float lerpf(float a, float b, float p) {
    return a + (b - a) * p;
}

__device__ float blerpf(float c00, float c10, float c01, float c11, float tx, float ty) {
    return lerpf(lerpf(c00, c10, tx), lerpf(c01, c11, tx), ty);
}

__device__ float fracf(float x) {
  return x - truncf(x);
}

#define EPSILON 0.000000001f


#ifdef ADD_FEATURE_FAST_NOISE
//--------------------------------- Noise (for supporting wfields) ----------------------------------
// partial CUDA-port of FastNoise: https://github.com/Auburn/FastNoise_Java
// restrictions:
//  - only 3d-noise is supported
// -  and NoiseLookup-return-type of cellular noise is not supported because it is very complicated to set up (at least on GPU, in comparison to all other types)
typedef enum {Value, ValueFractal, Perlin, PerlinFractal, Simplex, SimplexFractal, Cellular, WhiteNoise, Cubic, CubicFractal} NoiseType;
typedef enum {Linear, Hermite, Quintic} Interp;
typedef enum {FBM, Billow, RigidMulti} FractalType;
typedef enum {Euclidean, Manhattan, Natural} CellularDistanceFunction;
typedef enum {CellValue, Distance, Distance2, Distance2Add, Distance2Sub, Distance2Mul, Distance2Div} CellularReturnType;

typedef struct __align__(8)
{
  int m_seed; // seed used for all noise types
              // Default: 1337
  float m_frequency; // frequency for all noise types
                     // Default: 0.01
  Interp m_interp; // possible interpolation methods (lowest to highest quality): Linear, Hermite, Quintic
                   // used in Value, Gradient Noise and Position Perturbing
                   // Default: Quintic
  NoiseType m_noiseType; // Default: Simplex
  int m_octaves; // octave count for all fractal noise types
                 // Default: 3
  float m_lacunarity; // octave lacunarity for all fractal noise types
                      // Default: 2.0
  float m_gain; //  octave gain for all fractal noise types
               	// Default: 0.5
  FractalType m_fractalType; // method for combining octaves in all fractal noise types
                            // Default: FBM
  CellularDistanceFunction m_cellularDistanceFunction; 	// distance function used in cellular noise calculations
                                                       	// Default: Euclidean
  CellularReturnType m_cellularReturnType; 	// return type from cellular noise calculations
                                           	// Default: CellValue
  float m_fractalBounding;
} FastNoise;


__device__ void calculateFractalBounding(FastNoise* n) {
    float amp = n->m_gain;
    float ampFractal = 1;
    for (int i = 1; i < n->m_octaves; i++) {
        ampFractal += amp;
        amp *= n->m_gain;
    }
    n->m_fractalBounding = 1 / ampFractal;
}

__device__ void fastNoise_init(FastNoise* n) {
  n->m_seed = 1337;
  n->m_frequency = 0.01f;
  n->m_interp = Quintic;
  n->m_noiseType = Simplex;
  n->m_octaves = 3;
  n->m_lacunarity = 2.0f;
  n->m_gain = 0.5f;
  n->m_fractalType = FBM;
  n->m_cellularDistanceFunction = Euclidean;
  n->m_cellularReturnType = Distance;
  calculateFractalBounding(n);
}

__device__ void fastNoise_prepare(FastNoise* n) {
  calculateFractalBounding(n);
}

__device__ int fastFloor(float f) {
  return (f >= 0 ? (int) f : (int) f - 1);
}

__device__ int fastRound(float f) {
  return (f >= 0) ? (int) (f + (float) 0.5) : (int) (f - (float) 0.5);
}

__device__ float lerp(float a, float b, float t) {
  return a + t * (b - a);
}

__device__ float interpHermiteFunc(float t) {
  return t * t * (3 - 2 * t);
}

__device__ float interpQuinticFunc(float t) {
  return t * t * t * (t * (t * 6 - 15) + 10);
}

__device__ float cubicLerp(float a, float b, float c, float d, float t) {
  float p = (d - c) - (a - b);
  return t * t * t * p + t * t * ((a - b) - p) + t * (c - a) + b;
}

__device__ __constant__ float GRAD_3D_x[16] =  { 1, -1, 1, -1, 1, -1, 1, -1, 0, 0, 0, 0, 1, 0, -1, 0 };
__device__ __constant__ float GRAD_3D_y[16] =  { 1, 1, -1, -1, 0, 0, 0, 0, 1, -1, 1, -1, 1, -1, 1, -1 };
__device__ __constant__ float GRAD_3D_z[16] =  { 0, 0, 0, 0, 1, 1, -1, -1, 1, 1, -1, -1, 0, 1, 0, -1 };

// Hashing
__device__ __constant__ int X_PRIME = 1619;
__device__ __constant__ int Y_PRIME = 31337;
__device__ __constant__ int Z_PRIME = 6971;
__device__ __constant__ int W_PRIME = 1013;

__device__ int hash2D(int seed, int x, int y) {
    int hash = seed;
    hash ^= X_PRIME * x;
    hash ^= Y_PRIME * y;

    hash = hash * hash * hash * 60493;
    hash = (hash >> 13) ^ hash;

    return hash;
}

__device__ int hash3D(int seed, int x, int y, int z) {
    int hash = seed;
    hash ^= X_PRIME * x;
    hash ^= Y_PRIME * y;
    hash ^= Z_PRIME * z;

    hash = hash * hash * hash * 60493;
    hash = (hash >> 13) ^ hash;

    return hash;
}

__device__ int hash4D(int seed, int x, int y, int z, int w) {
    int hash = seed;
    hash ^= X_PRIME * x;
    hash ^= Y_PRIME * y;
    hash ^= Z_PRIME * z;
    hash ^= W_PRIME * w;

    hash = hash * hash * hash * 60493;
    hash = (hash >> 13) ^ hash;

    return hash;
}

__device__ float valCoord2D(int seed, int x, int y) {
    int n = seed;
    n ^= X_PRIME * x;
    n ^= Y_PRIME * y;

    return (n * n * n * 60493) / (float) 2147483648.0;
}

__device__ float valCoord3D(int seed, int x, int y, int z) {
    int n = seed;
    n ^= X_PRIME * x;
    n ^= Y_PRIME * y;
    n ^= Z_PRIME * z;

    return (n * n * n * 60493) / (float) 2147483648.0;
}

__device__ float valCoord4D(int seed, int x, int y, int z, int w) {
    int n = seed;
    n ^= X_PRIME * x;
    n ^= Y_PRIME * y;
    n ^= Z_PRIME * z;
    n ^= W_PRIME * w;

    return (n * n * n * 60493) / (float) 2147483648.0;
}

__device__ float gradCoord3D(int seed, int x, int y, int z, float xd, float yd, float zd) {
    int hash = seed;
    hash ^= X_PRIME * x;
    hash ^= Y_PRIME * y;
    hash ^= Z_PRIME * z;

    hash = hash * hash * hash * 60493;
    hash = (hash >> 13) ^ hash;

    int idx = hash & 15;

    return xd * GRAD_3D_x[idx] + yd * GRAD_3D_y[idx] + zd * GRAD_3D_z[idx];
}

__device__ float gradCoord4D(int seed, int x, int y, int z, int w, float xd, float yd, float zd, float wd) {
    int hash = seed;
    hash ^= X_PRIME * x;
    hash ^= Y_PRIME * y;
    hash ^= Z_PRIME * z;
    hash ^= W_PRIME * w;

    hash = hash * hash * hash * 60493;
    hash = (hash >> 13) ^ hash;

    hash &= 31;
    float a = yd, b = zd, c = wd;            // X,Y,Z
    switch (hash >> 3) {          // OR, DEPENDING ON HIGH ORDER 2 BITS:
        case 1:
            a = wd;
            b = xd;
            c = yd;
            break;     // W,X,Y
        case 2:
            a = zd;
            b = wd;
            c = xd;
            break;     // Z,W,X
        case 3:
            a = yd;
            b = zd;
            c = wd;
            break;     // Y,Z,W
    }
    return ((hash & 4) == 0 ? -a : a) + ((hash & 2) == 0 ? -b : b) + ((hash & 1) == 0 ? -c : c);
}

// White Noise
#ifdef ADD_FEATURE_WHITE_NOISE
__device__ int floatToIntBits(float  x)
{
  union {
    float f;  // assuming 32-bit IEEE 754 single-precision
    int i;    // assuming 32-bit 2's complement int
  } u;

  if (isnan(x)) {
    return 0x7fc00000;
  } else {
    u.f = x;
    return u.i;
  }
}

__device__  int floatCast2Int(float f) {
    int i = floatToIntBits(f);
    return i ^ (i >> 16);
}
#endif // ADD_FEATURE_WHITE_NOISE

#ifdef ADD_FEATURE_WHITE_NOISE
__device__ float getWhiteNoise(FastNoise* n, float x, float y, float z) {
    int xi = floatCast2Int(x);
    int yi = floatCast2Int(y);
    int zi = floatCast2Int(z);

    return valCoord3D(n->m_seed, xi, yi, zi);
}

__device__ float getWhiteNoiseInt(FastNoise* n, int x, int y, int z) {
    return valCoord3D(n->m_seed, x, y, z);
}
#endif // ADD_FEATURE_WHITE_NOISE

// Value Noise
#ifdef ADD_FEATURE_VALUE_NOISE
__device__ float singleValue(FastNoise* n,int seed, float x, float y, float z) {
    int x0 = fastFloor(x);
    int y0 = fastFloor(y);
    int z0 = fastFloor(z);
    int x1 = x0 + 1;
    int y1 = y0 + 1;
    int z1 = z0 + 1;

    float xs, ys, zs;
    switch (n->m_interp) {
        default:
        case Linear:
            xs = x - x0;
            ys = y - y0;
            zs = z - z0;
            break;
        case Hermite:
            xs = interpHermiteFunc(x - x0);
            ys = interpHermiteFunc(y - y0);
            zs = interpHermiteFunc(z - z0);
            break;
        case Quintic:
            xs = interpQuinticFunc(x - x0);
            ys = interpQuinticFunc(y - y0);
            zs = interpQuinticFunc(z - z0);
            break;
    }

    float xf00 = lerp(valCoord3D(seed, x0, y0, z0), valCoord3D(seed, x1, y0, z0), xs);
    float xf10 = lerp(valCoord3D(seed, x0, y1, z0), valCoord3D(seed, x1, y1, z0), xs);
    float xf01 = lerp(valCoord3D(seed, x0, y0, z1), valCoord3D(seed, x1, y0, z1), xs);
    float xf11 = lerp(valCoord3D(seed, x0, y1, z1), valCoord3D(seed, x1, y1, z1), xs);

    float yf0 = lerp(xf00, xf10, ys);
    float yf1 = lerp(xf01, xf11, ys);

    return lerp(yf0, yf1, zs);
}

__device__ float singleValueFractalFBM(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = singleValue(n, seed, x, y, z);
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += singleValue(n, ++seed, x, y, z) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singleValueFractalBillow(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = fabsf(singleValue(n, seed, x, y, z)) * 2 - 1;
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += (fabsf(singleValue(n, ++seed, x, y, z)) * 2 - 1) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singleValueFractalRigidMulti(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = 1 - fabsf(singleValue(n, seed, x, y, z));
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum -= (1 - fabsf(singleValue(n, ++seed, x, y, z))) * amp;
    }

    return sum;
}

__device__ float getValue(FastNoise* n, float x, float y, float z) {
    return singleValue(n, n->m_seed, x * n->m_frequency, y * n->m_frequency, z * n->m_frequency);
}

__device__ float getValueFractal(FastNoise* n, float x, float y, float z) {
    x *= n->m_frequency;
    y *= n->m_frequency;
    z *= n->m_frequency;

    switch (n->m_fractalType) {
        case FBM:
            return singleValueFractalFBM(n, x, y, z);
        case Billow:
            return singleValueFractalBillow(n, x, y, z);
        case RigidMulti:
            return singleValueFractalRigidMulti(n, x, y, z);
        default:
            return 0;
    }
}
#endif // ADD_FEATURE_VALUE_NOISE

#ifdef ADD_FEATURE_PERLIN_NOISE
// Perlin Noise
__device__ float singlePerlin(FastNoise* n, int seed, float x, float y, float z) {
    int x0 = fastFloor(x);
    int y0 = fastFloor(y);
    int z0 = fastFloor(z);
    int x1 = x0 + 1;
    int y1 = y0 + 1;
    int z1 = z0 + 1;

    float xs, ys, zs;
    switch (n->m_interp) {
        default:
        case Linear:
            xs = x - x0;
            ys = y - y0;
            zs = z - z0;
            break;
        case Hermite:
            xs = interpHermiteFunc(x - x0);
            ys = interpHermiteFunc(y - y0);
            zs = interpHermiteFunc(z - z0);
            break;
        case Quintic:
            xs = interpQuinticFunc(x - x0);
            ys = interpQuinticFunc(y - y0);
            zs = interpQuinticFunc(z - z0);
            break;
    }

    float xd0 = x - x0;
    float yd0 = y - y0;
    float zd0 = z - z0;
    float xd1 = xd0 - 1;
    float yd1 = yd0 - 1;
    float zd1 = zd0 - 1;

    float xf00 = lerp(gradCoord3D(seed, x0, y0, z0, xd0, yd0, zd0), gradCoord3D(seed, x1, y0, z0, xd1, yd0, zd0), xs);
    float xf10 = lerp(gradCoord3D(seed, x0, y1, z0, xd0, yd1, zd0), gradCoord3D(seed, x1, y1, z0, xd1, yd1, zd0), xs);
    float xf01 = lerp(gradCoord3D(seed, x0, y0, z1, xd0, yd0, zd1), gradCoord3D(seed, x1, y0, z1, xd1, yd0, zd1), xs);
    float xf11 = lerp(gradCoord3D(seed, x0, y1, z1, xd0, yd1, zd1), gradCoord3D(seed, x1, y1, z1, xd1, yd1, zd1), xs);

    float yf0 = lerp(xf00, xf10, ys);
    float yf1 = lerp(xf01, xf11, ys);

    return lerp(yf0, yf1, zs);
}

__device__ float getPerlin(FastNoise* n, float x, float y, float z) {
    return singlePerlin(n, n->m_seed, x * n->m_frequency, y * n->m_frequency, z * n->m_frequency);
}

__device__ float singlePerlinFractalFBM(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = singlePerlin(n, seed, x, y, z);
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += singlePerlin(n, ++seed, x, y, z) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singlePerlinFractalBillow(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = fabsf(singlePerlin(n, seed, x, y, z)) * 2 - 1;
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += (fabsf(singlePerlin(n, ++seed, x, y, z)) * 2 - 1) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singlePerlinFractalRigidMulti(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = 1 - fabsf(singlePerlin(n, seed, x, y, z));
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum -= (1 - fabsf(singlePerlin(n, ++seed, x, y, z))) * amp;
    }

    return sum;
}

__device__ float getPerlinFractal(FastNoise* n, float x, float y, float z) {
    x *= n->m_frequency;
    y *= n->m_frequency;
    z *= n->m_frequency;

    switch (n->m_fractalType) {
        case FBM:
            return singlePerlinFractalFBM(n, x, y, z);
        case Billow:
            return singlePerlinFractalBillow(n, x, y, z);
        case RigidMulti:
            return singlePerlinFractalRigidMulti(n, x, y, z);
        default:
            return 0;
    }
}
#endif // ADD_FEATURE_PERLIN_NOISE

// Simplex Noise
#ifdef ADD_FEATURE_SIMPLEX_NOISE
__device__ __constant__ float F3 = (float) (1.0 / 3.0);
__device__ __constant__ float G3 = (float) (1.0 / 6.0);
__device__ __constant__ float G33 =(float) ((1.0 / 6.0) * 3 - 1);
#endif // ADD_FEATURE_SIMPLEX_NOISE

#ifdef ADD_FEATURE_SIMPLEX_NOISE
__device__ float singleSimplex(int seed, float x, float y, float z) {
    float t = (x + y + z) * F3;
    int i = fastFloor(x + t);
    int j = fastFloor(y + t);
    int k = fastFloor(z + t);

    t = (i + j + k) * G3;
    float x0 = x - (i - t);
    float y0 = y - (j - t);
    float z0 = z - (k - t);

    int i1, j1, k1;
    int i2, j2, k2;

    if (x0 >= y0) {
        if (y0 >= z0) {
            i1 = 1;
            j1 = 0;
            k1 = 0;
            i2 = 1;
            j2 = 1;
            k2 = 0;
        } else if (x0 >= z0) {
            i1 = 1;
            j1 = 0;
            k1 = 0;
            i2 = 1;
            j2 = 0;
            k2 = 1;
        } else // x0 < z0
        {
            i1 = 0;
            j1 = 0;
            k1 = 1;
            i2 = 1;
            j2 = 0;
            k2 = 1;
        }
    } else // x0 < y0
    {
        if (y0 < z0) {
            i1 = 0;
            j1 = 0;
            k1 = 1;
            i2 = 0;
            j2 = 1;
            k2 = 1;
        } else if (x0 < z0) {
            i1 = 0;
            j1 = 1;
            k1 = 0;
            i2 = 0;
            j2 = 1;
            k2 = 1;
        } else // x0 >= z0
        {
            i1 = 0;
            j1 = 1;
            k1 = 0;
            i2 = 1;
            j2 = 1;
            k2 = 0;
        }
    }

    float x1 = x0 - i1 + G3;
    float y1 = y0 - j1 + G3;
    float z1 = z0 - k1 + G3;
    float x2 = x0 - i2 + F3;
    float y2 = y0 - j2 + F3;
    float z2 = z0 - k2 + F3;
    float x3 = x0 + G33;
    float y3 = y0 + G33;
    float z3 = z0 + G33;

    float n0, n1, n2, n3;

    t = (float) 0.6 - x0 * x0 - y0 * y0 - z0 * z0;
    if (t < 0) n0 = 0;
    else {
        t *= t;
        n0 = t * t * gradCoord3D(seed, i, j, k, x0, y0, z0);
    }

    t = (float) 0.6 - x1 * x1 - y1 * y1 - z1 * z1;
    if (t < 0) n1 = 0;
    else {
        t *= t;
        n1 = t * t * gradCoord3D(seed, i + i1, j + j1, k + k1, x1, y1, z1);
    }

    t = (float) 0.6 - x2 * x2 - y2 * y2 - z2 * z2;
    if (t < 0) n2 = 0;
    else {
        t *= t;
        n2 = t * t * gradCoord3D(seed, i + i2, j + j2, k + k2, x2, y2, z2);
    }

    t = (float) 0.6 - x3 * x3 - y3 * y3 - z3 * z3;
    if (t < 0) n3 = 0;
    else {
        t *= t;
        n3 = t * t * gradCoord3D(seed, i + 1, j + 1, k + 1, x3, y3, z3);
    }

    return 32 * (n0 + n1 + n2 + n3);
}

__device__ float getSimplex(FastNoise* n, float x, float y, float z) {
    return singleSimplex(n->m_seed, x * n->m_frequency, y * n->m_frequency, z * n->m_frequency);
}

__device__ float singleSimplexFractalFBM(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = singleSimplex(seed, x, y, z);
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += singleSimplex(++seed, x, y, z) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singleSimplexFractalBillow(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = fabsf(singleSimplex(seed, x, y, z)) * 2 - 1;
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += (fabsf(singleSimplex(++seed, x, y, z)) * 2 - 1) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singleSimplexFractalRigidMulti(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = 1 - fabsf(singleSimplex(seed, x, y, z));
    float amp = 1;

    for (int i = 1; i < n->m_octaves; i++) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum -= (1 - fabsf(singleSimplex(++seed, x, y, z))) * amp;
    }

    return sum;
}

__device__ float getSimplexFractal(FastNoise* n, float x, float y, float z) {
    x *= n->m_frequency;
    y *= n->m_frequency;
    z *= n->m_frequency;

    switch (n->m_fractalType) {
        case FBM:
            return singleSimplexFractalFBM(n, x, y, z);
        case Billow:
            return singleSimplexFractalBillow(n, x, y, z);
        case RigidMulti:
            return singleSimplexFractalRigidMulti(n, x, y, z);
        default:
            return 0;
    }
}
#endif // ADD_FEATURE_SIMPLEX_NOISE

#ifdef ADD_FEATURE_SIMPLEX_NOISE
__device__ __constant__ float F2 = (float) (1.0 / 2.0);
__device__ __constant__ float G2 = (float) (1.0 / 4.0);
#endif

// Cubic Noise
#ifdef ADD_FEATURE_CUBIC_NOISE
__device__ __constant__ float CUBIC_3D_BOUNDING = 1 / (float) (1.5 * 1.5 * 1.5);

__device__ float singleCubic(FastNoise* n, int seed, float x, float y, float z) {
    int x1 = fastFloor(x);
    int y1 = fastFloor(y);
    int z1 = fastFloor(z);

    int x0 = x1 - 1;
    int y0 = y1 - 1;
    int z0 = z1 - 1;
    int x2 = x1 + 1;
    int y2 = y1 + 1;
    int z2 = z1 + 1;
    int x3 = x1 + 2;
    int y3 = y1 + 2;
    int z3 = z1 + 2;

    float xs = x - (float) x1;
    float ys = y - (float) y1;
    float zs = z - (float) z1;

    return cubicLerp(
        cubicLerp(
            cubicLerp(valCoord3D(seed, x0, y0, z0), valCoord3D(seed, x1, y0, z0), valCoord3D(seed, x2, y0, z0), valCoord3D(seed, x3, y0, z0), xs),
            cubicLerp(valCoord3D(seed, x0, y1, z0), valCoord3D(seed, x1, y1, z0), valCoord3D(seed, x2, y1, z0), valCoord3D(seed, x3, y1, z0), xs),
            cubicLerp(valCoord3D(seed, x0, y2, z0), valCoord3D(seed, x1, y2, z0), valCoord3D(seed, x2, y2, z0), valCoord3D(seed, x3, y2, z0), xs),
            cubicLerp(valCoord3D(seed, x0, y3, z0), valCoord3D(seed, x1, y3, z0), valCoord3D(seed, x2, y3, z0), valCoord3D(seed, x3, y3, z0), xs),
            ys),
        cubicLerp(
            cubicLerp(valCoord3D(seed, x0, y0, z1), valCoord3D(seed, x1, y0, z1), valCoord3D(seed, x2, y0, z1), valCoord3D(seed, x3, y0, z1), xs),
            cubicLerp(valCoord3D(seed, x0, y1, z1), valCoord3D(seed, x1, y1, z1), valCoord3D(seed, x2, y1, z1), valCoord3D(seed, x3, y1, z1), xs),
            cubicLerp(valCoord3D(seed, x0, y2, z1), valCoord3D(seed, x1, y2, z1), valCoord3D(seed, x2, y2, z1), valCoord3D(seed, x3, y2, z1), xs),
            cubicLerp(valCoord3D(seed, x0, y3, z1), valCoord3D(seed, x1, y3, z1), valCoord3D(seed, x2, y3, z1), valCoord3D(seed, x3, y3, z1), xs),
            ys),
        cubicLerp(
            cubicLerp(valCoord3D(seed, x0, y0, z2), valCoord3D(seed, x1, y0, z2), valCoord3D(seed, x2, y0, z2), valCoord3D(seed, x3, y0, z2), xs),
            cubicLerp(valCoord3D(seed, x0, y1, z2), valCoord3D(seed, x1, y1, z2), valCoord3D(seed, x2, y1, z2), valCoord3D(seed, x3, y1, z2), xs),
            cubicLerp(valCoord3D(seed, x0, y2, z2), valCoord3D(seed, x1, y2, z2), valCoord3D(seed, x2, y2, z2), valCoord3D(seed, x3, y2, z2), xs),
            cubicLerp(valCoord3D(seed, x0, y3, z2), valCoord3D(seed, x1, y3, z2), valCoord3D(seed, x2, y3, z2), valCoord3D(seed, x3, y3, z2), xs),
            ys),
        cubicLerp(
            cubicLerp(valCoord3D(seed, x0, y0, z3), valCoord3D(seed, x1, y0, z3), valCoord3D(seed, x2, y0, z3), valCoord3D(seed, x3, y0, z3), xs),
            cubicLerp(valCoord3D(seed, x0, y1, z3), valCoord3D(seed, x1, y1, z3), valCoord3D(seed, x2, y1, z3), valCoord3D(seed, x3, y1, z3), xs),
            cubicLerp(valCoord3D(seed, x0, y2, z3), valCoord3D(seed, x1, y2, z3), valCoord3D(seed, x2, y2, z3), valCoord3D(seed, x3, y2, z3), xs),
            cubicLerp(valCoord3D(seed, x0, y3, z3), valCoord3D(seed, x1, y3, z3), valCoord3D(seed, x2, y3, z3), valCoord3D(seed, x3, y3, z3), xs),
            ys),
        zs) * CUBIC_3D_BOUNDING;
}


__device__ float getCubic(FastNoise* n, float x, float y, float z) {
    return singleCubic(n, n->m_seed, x * n->m_frequency, y * n->m_frequency, z * n->m_frequency);
}

__device__ float singleCubicFractalFBM(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = singleCubic(n, seed, x, y, z);
    float amp = 1;
    int i = 0;

    while (++i < n->m_octaves) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += singleCubic(n, ++seed, x, y, z) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singleCubicFractalBillow(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = fabsf(singleCubic(n, seed, x, y, z)) * 2 - 1;
    float amp = 1;
    int i = 0;

    while (++i < n->m_octaves) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum += (fabsf(singleCubic(n, ++seed, x, y, z)) * 2 - 1) * amp;
    }

    return sum * n->m_fractalBounding;
}

__device__ float singleCubicFractalRigidMulti(FastNoise* n, float x, float y, float z) {
    int seed = n->m_seed;
    float sum = 1 - fabsf(singleCubic(n, seed, x, y, z));
    float amp = 1;
    int i = 0;

    while (++i < n->m_octaves) {
        x *= n->m_lacunarity;
        y *= n->m_lacunarity;
        z *= n->m_lacunarity;

        amp *= n->m_gain;
        sum -= (1 - fabsf(singleCubic(n, ++seed, x, y, z))) * amp;
    }

    return sum;
}

__device__ float getCubicFractal(FastNoise* n, float x, float y, float z) {
    x *= n->m_frequency;
    y *= n->m_frequency;
    z *= n->m_frequency;

    switch (n->m_fractalType) {
        case FBM:
            return singleCubicFractalFBM(n, x, y, z);
        case Billow:
            return singleCubicFractalBillow(n, x, y, z);
        case RigidMulti:
            return singleCubicFractalRigidMulti(n, x, y, z);
        default:
            return 0;
    }
}
#endif // ADD_FEATURE_CUBIC_NOISE

// Cellular Noise
#ifdef ADD_FEATURE_CELLULAR_NOISE
__device__ __constant__ float CELL_3D_x[] =  {
    0.1453787434f, -0.01242829687f, 0.2877979582f, -0.07732986802f, 0.1107205875f, 0.2755209141f, 0.294168941f, 0.4000921098f,
    -0.1697304074f, -0.1483224484f, 0.2623596946f, -0.2709003183f, -0.03516550699f, -0.1267712655f, 0.02952021915f, -0.2806854217f,
    -0.171159547f, 0.2113227183f, -0.1024352839f, -0.3304249877f, 0.2091111325f, 0.344678154f, 0.1984478035f, -0.2929008603f,
    -0.1617332831f, -0.3582060271f, -0.1852067326f, 0.3046301062f, -0.03816768434f, -0.4084952196f, -0.02687443361f, -0.03801098351f,
    0.2371120802f, 0.4447660503f, 0.01985147278f, 0.4274339143f, -0.2072988631f, -0.3791240978f, -0.2098721267f, 0.01582798878f,
    -0.1888129464f, 0.1612988974f, -0.08974491322f, 0.07041229526f, -0.1082925611f, 0.2474100658f, -0.1068836661f, 0.2396452163f,
    -0.3063886072f, 0.1593342891f, 0.2709690528f, -0.1519780427f, 0.1699773681f, -0.1986155616f, -0.1887482106f, 0.2659103394f,
    -0.08838976154f, -0.04201869311f, -0.3230334656f, 0.2612720941f, 0.385713046f, 0.07654967953f, 0.4317038818f, -0.2890436293f,
    -0.2201947582f, 0.4161322773f, 0.2204718095f, -0.1040307469f, -0.1432122615f, 0.3978380468f, -0.2599274663f, 0.4032618332f,
    -0.08953470255f, 0.118937202f, 0.02167047076f, -0.3411343612f, 0.3162964612f, 0.2355138889f, -0.02874541518f, -0.2461455173f,
    0.04208029445f, 0.2727458746f, -0.1347522818f, 0.3829624424f, -0.3547613644f, 0.2305790207f, -0.08323845599f, 0.2993663085f,
    -0.2154865723f, 0.01683355354f, 0.05240429123f, 0.00940104872f, 0.3465688735f, -0.3706867948f, 0.2741169781f, 0.06413433865f,
    -0.388187972f, 0.06419469312f, -0.1986120739f, -0.203203009f, -0.1389736354f, -0.06555641638f, -0.2529246486f, 0.1444476522f,
    -0.3643780054f, 0.4286142488f, 0.165872923f, 0.2219610524f, 0.04322940318f, -0.08481269795f, 0.1822082075f, -0.3269323334f,
    -0.4080485344f, 0.2676025294f, 0.3024892441f, 0.1448494052f, 0.4198402157f, -0.3008872161f, 0.3639310428f, 0.3295806598f,
    0.2776259487f, 0.4149000507f, 0.145016715f, 0.09299023471f, 0.1028907093f, 0.2683057049f, -0.4227307273f, -0.1781224702f,
    0.4390788626f, 0.2972583585f, -0.1707002821f, 0.3806686614f, -0.1751445661f, -0.2227237566f, 0.1369633021f, -0.3529503428f,
    -0.2590744185f, -0.3784019401f, -0.05635805671f, 0.3251428613f, -0.4190995804f, -0.3253150961f, 0.2857945863f, -0.2733604046f,
    0.219003657f, 0.3182767252f, -0.03222023115f, -0.3087780231f, -0.06487611647f, 0.3921171432f, -0.1606404506f, -0.03767771199f,
    0.1394866832f, -0.4345093872f, -0.1044637494f, 0.2658727501f, 0.2051461999f, -0.266085566f, 0.07849405464f, -0.2160686338f,
    -0.185779186f, 0.02492421743f, -0.120167831f, -0.02160084693f, 0.2597670064f, -0.1611553854f, -0.3278896792f, 0.2822734956f,
    0.03169341113f, 0.2202613604f, 0.2933396046f, -0.3194922995f, -0.3441586045f, 0.2703645948f, 0.2298568861f, 0.09326603877f,
    -0.1116165319f, 0.2172907365f, 0.1991339479f, -0.0541918155f, 0.08871336998f, 0.2787673278f, -0.322166438f, -0.4277366384f,
    0.240131882f, 0.1448607981f, -0.3837065682f, -0.4382627882f, -0.37728353f, 0.1259579313f, -0.1406285511f, -0.1580694418f,
    0.2477612106f, 0.2916132853f, 0.07365265219f, -0.26126526f, -0.3721862032f, -0.3691191571f, 0.2278441737f, 0.363398169f,
    -0.304231482f, -0.3199312232f, 0.2874852279f, -0.1451096801f, 0.3220090754f, -0.1247400865f, -0.2829555867f, 0.1069384374f,
    -0.1420661144f, -0.250548338f, 0.3265787872f, 0.07646097258f, 0.3451771584f, 0.298137964f, 0.2812250376f, 0.4390345476f,
    0.2148373234f, 0.2595421179f, 0.3182823114f, -0.4089859285f, -0.2826749061f, 0.3483864637f, -0.3226415069f, 0.4330734858f,
    -0.08717822568f, -0.2149678299f, -0.2687330705f, 0.2105665099f, 0.4361845915f, 0.05333333359f, -0.05986216652f, 0.3664988455f,
    -0.2341015558f, -0.04730947785f, -0.2391566239f, -0.1242081035f, 0.2614832715f, -0.2728794681f, 0.007892900508f, -0.01730330376f,
    0.2054835762f, -0.3231994983f, -0.2669545963f, -0.05554372779f, -0.2083935713f, 0.06989323478f, 0.3847566193f, -0.3026215288f,
    0.3450735512f, 0.1814473292f, -0.03855010448f, 0.3533670318f, -0.007945601311f, 0.4063099273f, -0.2016773589f, -0.07527055435f,
};

__device__ __constant__ float CELL_3D_y[] = {
    -0.4149781685f, -0.1457918398f, -0.02606483451f, 0.2377094325f, -0.3552302079f, 0.2640521179f, 0.1526064594f, -0.2034056362f,
    0.3970864695f, -0.3859694688f, -0.2354852944f, 0.3505271138f, 0.3885234328f, 0.1920044036f, 0.4409685861f, -0.266996757f,
    0.2141185563f, 0.3902405947f, 0.2128044156f, -0.1566986703f, 0.3133278055f, -0.1944240454f, -0.3214342325f, 0.2262915116f,
    0.006314769776f, -0.148303178f, -0.3454119342f, 0.1026310383f, -0.2551766358f, 0.1805950793f, -0.2749741471f, 0.3277859044f,
    0.2900386767f, 0.03946930643f, -0.01503183293f, 0.03345994256f, 0.2871414597f, 0.1281177671f, -0.1007087278f, 0.4263894424f,
    -0.3160996813f, -0.1974805082f, 0.229148752f, 0.4150230285f, -0.1586061639f, -0.3309414609f, -0.2701644537f, 0.06803600538f,
    0.2597428179f, -0.3114350249f, 0.1412648683f, 0.3623355133f, 0.3456012883f, 0.3836276443f, -0.2050154888f, 0.3015631259f,
    -0.4288819642f, 0.3099592485f, 0.201549922f, 0.2759854499f, 0.2193460345f, 0.3721732183f, -0.02577753072f, -0.3418179959f,
    0.383023377f, -0.1669634289f, 0.02654238946f, 0.3890079625f, 0.371614387f, -0.06206669342f, 0.2616724959f, -0.1124593585f,
    -0.3048244735f, -0.2875221847f, -0.03284630549f, 0.2500031105f, 0.3082064153f, -0.3439334267f, -0.3955933019f, 0.02020282325f,
    -0.4470439576f, 0.2288471896f, -0.02720848277f, 0.1231931484f, 0.1271702173f, 0.3063895591f, -0.1922245118f, -0.2619918095f,
    0.2706747713f, -0.2680655787f, 0.4335128183f, -0.4472890582f, 0.01141914583f, -0.2551104378f, 0.2139972417f, 0.1708718512f,
    -0.03973280434f, -0.2803682491f, -0.3391173584f, -0.3871641506f, -0.2775901578f, 0.342253257f, -0.2904227915f, 0.1069184044f,
    -0.2447099973f, -0.1358496089f, -0.3136808464f, -0.3658139958f, -0.3832730794f, -0.4404869674f, -0.3953259299f, 0.3036542563f,
    0.04227858267f, -0.01299671652f, -0.1009990293f, 0.425921681f, 0.08062320474f, -0.333040905f, -0.1291284382f, 0.0184175994f,
    -0.2974929052f, -0.144793182f, -0.0398992945f, -0.299732164f, -0.361266869f, -0.07076041213f, -0.07933161816f, 0.1806857196f,
    -0.02841848598f, 0.2382799621f, 0.2215845691f, 0.1471852559f, -0.274887877f, -0.2316778837f, 0.1341343041f, -0.2472893463f,
    -0.2985577559f, 0.2199816631f, 0.1485737441f, 0.09666046873f, 0.1406751354f, -0.3080335042f, -0.05796152095f, 0.1973770973f,
    0.2410037886f, -0.271342949f, -0.3331161506f, 0.1992794134f, -0.4311322747f, -0.06294284106f, -0.358928121f, -0.2290351443f,
    -0.3602213994f, 0.005751117145f, 0.4168128432f, 0.2551943237f, 0.1975390727f, 0.23483312f, -0.3300346342f, 0.05376451292f,
    0.2148499206f, -0.3229954284f, 0.4017266681f, -0.06885389554f, 0.3096300784f, -0.09823036005f, 0.1461670309f, 0.03754421121f,
    0.347405252f, -0.3460788041f, 0.3031973659f, 0.2453752201f, -0.1698856132f, -0.3574277231f, 0.3744156221f, -0.3170108894f,
    -0.2985018719f, -0.3460005203f, 0.3820341668f, -0.2103145071f, 0.2012117383f, 0.3505404674f, 0.3067213525f, 0.132066775f,
    -0.1612516055f, -0.2387819045f, -0.2206398454f, -0.09082753406f, 0.05445141085f, 0.348394558f, -0.270877371f, 0.4162931958f,
    -0.2927867412f, 0.3312535401f, -0.1666159848f, -0.2422237692f, 0.252790166f, -0.255281188f, -0.3358364886f, -0.2310190248f,
    -0.2698452035f, 0.316332536f, 0.1642275508f, 0.3277541114f, 0.0511344108f, -0.04333605335f, -0.3056190617f, 0.3491024667f,
    -0.3055376754f, 0.3156466809f, 0.1871229129f, -0.3026690852f, 0.2757120714f, 0.2852657134f, 0.3466716415f, -0.09790429955f,
    0.1850172527f, -0.07946825393f, -0.307355516f, -0.04647718411f, 0.07417482322f, 0.225442246f, -0.1420585388f, -0.118868561f,
    -0.3909896417f, 0.3939973956f, 0.322686276f, -0.1961317136f, -0.1105517485f, -0.313639498f, 0.1361029153f, 0.2550543014f,
    -0.182405731f, -0.4222150243f, -0.2577696514f, 0.4256953395f, -0.3650179274f, -0.3499628774f, -0.1672771315f, 0.2978486637f,
    -0.3252600376f, 0.1564282844f, 0.2599343665f, 0.3170813944f, -0.310922837f, -0.3156141536f, -0.1605309138f, -0.3001537679f,
    0.08611519592f, -0.2788782453f, 0.09795110726f, 0.2665752752f, 0.140359426f, -0.1491768253f, 0.008816271194f, -0.425643481f,
};

__device__ __constant__ float CELL_3D_z[] = {
    -0.0956981749f, -0.4255470325f, -0.3449535616f, 0.3741848704f, -0.2530858567f, -0.238463215f, 0.3044271714f, 0.03244149937f,
    -0.1265461359f, 0.1775613147f, 0.2796677792f, -0.07901746678f, 0.2243054374f, 0.3867342179f, 0.08470692262f, 0.2289725438f,
    0.3568720405f, -0.07453178509f, -0.3830421561f, 0.2622305365f, -0.2461670583f, -0.2142341261f, -0.2445373252f, 0.2559320961f,
    -0.4198838754f, -0.2284613961f, -0.2211087107f, 0.314908508f, -0.3686842991f, 0.05492788837f, 0.3551999201f, 0.3059600725f,
    -0.2493099024f, 0.05590469027f, -0.4493105419f, -0.1366772882f, -0.2776273824f, 0.2057929936f, -0.3851122467f, 0.1429738373f,
    -0.2587096108f, -0.3707885038f, -0.3767448739f, -0.1590534329f, 0.4069604477f, 0.1782302128f, -0.3436379634f, -0.3747549496f,
    0.2028785103f, -0.2830561951f, -0.3303331794f, 0.2193527988f, 0.2327390037f, -0.1260225743f, -0.353330953f, -0.2021172246f,
    -0.1036702021f, 0.3235115047f, -0.2398478873f, -0.2409749453f, 0.07491837764f, 0.241095919f, 0.1243675091f, -0.04598084447f,
    -0.08548310451f, -0.03817251927f, -0.391391981f, -0.2008741118f, -0.2095065525f, 0.2009293758f, -0.2578084893f, 0.1650235939f,
    0.3186935478f, 0.325092195f, -0.4482761547f, 0.1537068389f, -0.08640228117f, -0.1695376245f, 0.2125550295f, -0.3761704803f,
    0.02968078139f, -0.2752065618f, -0.4284874806f, -0.2016512234f, 0.2459107769f, 0.2354968222f, 0.3982726409f, -0.2103333191f,
    0.287751117f, -0.3610505186f, -0.1087217856f, 0.04841609928f, -0.2868093776f, 0.003156692623f, -0.2855959784f, 0.4113266307f,
    -0.2241236325f, 0.3460819069f, 0.2192091725f, 0.1063600375f, -0.3257760473f, -0.2847192729f, 0.2327739768f, 0.4125570634f,
    -0.09922543227f, -0.01829506817f, -0.2767498872f, 0.1393320198f, 0.2318037215f, -0.03574965489f, 0.1140946023f, 0.05838957105f,
    -0.184956522f, 0.36155217f, -0.3174892964f, -0.0104580805f, 0.1404780841f, -0.03241355801f, -0.2310412139f, -0.3058388149f,
    -0.1921504723f, -0.09691688386f, 0.4241205002f, -0.3225111565f, 0.247789732f, -0.3542668666f, -0.1323073187f, -0.3716517945f,
    -0.09435116353f, -0.2394997452f, 0.3525077196f, -0.1895464869f, 0.3102596268f, 0.3149912482f, -0.4071228836f, -0.129514612f,
    -0.2150435121f, -0.1044989934f, 0.4210102279f, -0.2957006485f, -0.08405978803f, -0.04225456877f, 0.3427271751f, -0.2980207554f,
    -0.3105713639f, 0.1660509868f, -0.300824678f, -0.2596995338f, 0.1114273361f, -0.2116183942f, -0.2187812825f, 0.3855169162f,
    0.2308332918f, 0.1169124335f, -0.1336202785f, 0.2582393035f, 0.3484154868f, 0.2766800993f, -0.2956616708f, -0.3910546287f,
    0.3490352499f, -0.3123343347f, 0.1633259825f, 0.4441762538f, 0.1978643903f, 0.4085091653f, 0.2713366126f, -0.3484423997f,
    -0.2842624114f, -0.1849713341f, 0.1565989581f, -0.200538455f, -0.2349334659f, 0.04060059933f, 0.0973588921f, 0.3054595587f,
    0.3177080142f, -0.1885958001f, -0.1299829458f, 0.39412061f, 0.3926114802f, 0.04370535101f, 0.06804996813f, 0.04582286686f,
    0.344723946f, 0.3528435224f, 0.08116235683f, -0.04664855374f, 0.2391488697f, 0.2554522098f, -0.3306796947f, -0.06491553533f,
    -0.2353514536f, 0.08793624968f, 0.411478311f, 0.2748965434f, 0.008634938242f, 0.03290232422f, 0.1944244981f, 0.1306597909f,
    0.1926830856f, -0.008816977938f, -0.304764754f, -0.2720669462f, 0.3101538769f, -0.4301882115f, -0.1703910946f, -0.2630430352f,
    -0.2982682484f, -0.2002316239f, 0.2466400438f, 0.324106687f, -0.0856480183f, 0.179547284f, 0.05684409612f, -0.01278335452f,
    0.3494474791f, 0.3589187731f, -0.08203022006f, 0.1818526372f, 0.3421885344f, -0.1740766085f, -0.2796816575f, -0.02859407492f,
    -0.2050050172f, -0.03247898316f, -0.1617284888f, -0.3459683451f, 0.004616608544f, -0.3182543336f, -0.4247264031f, -0.05590974511f,
    0.3382670703f, -0.1483114513f, -0.2808182972f, -0.07652336246f, 0.02980623099f, 0.07458404908f, 0.4176793787f, -0.3368779738f,
    -0.2334146693f, -0.2712420987f, -0.2523278991f, -0.3144428146f, -0.2497981362f, 0.3130537363f, -0.1693876312f, -0.1443188342f,
    0.2756962409f, -0.3029914042f, 0.4375151083f, 0.08105160988f, -0.4274764309f, -0.1231199324f, -0.4021797064f, -0.1251477955f,
};

#define MAX_CELL_POS 32000
#define MIN_CELL_POS -32000

__device__ float singleCellular(FastNoise* n, float x, float y, float z) {
    int xr = max(min(fastRound(x), MAX_CELL_POS), MIN_CELL_POS);
    int yr = max(min(fastRound(y), MAX_CELL_POS), MIN_CELL_POS);
    int zr = max(min(fastRound(z), MAX_CELL_POS), MIN_CELL_POS);

    float distance = 999999;
    int xc = 0, yc = 0, zc = 0;

    switch (n->m_cellularDistanceFunction) {
        case Euclidean:
            for (int xi = xr - 1; xi <= xr + 1; xi++) {
                for (int yi = yr - 1; yi <= yr + 1; yi++) {
                    for (int zi = zr - 1; zi <= zr + 1; zi++) {
                        int idx = hash3D(n->m_seed, xi, yi, zi) & 255;


                        float vecX = xi - x + CELL_3D_x[idx];
                        float vecY = yi - y + CELL_3D_y[idx];
                        float vecZ = zi - z + CELL_3D_z[idx];

                        float newDistance = vecX * vecX + vecY * vecY + vecZ * vecZ;

                        if (newDistance < distance) {
                            distance = newDistance;
                            xc = xi;
                            yc = yi;
                            zc = zi;
                        }
                    }
                }
            }
            break;
        case Manhattan:
            for (int xi = xr - 1; xi <= xr + 1; xi++) {
                for (int yi = yr - 1; yi <= yr + 1; yi++) {
                    for (int zi = zr - 1; zi <= zr + 1; zi++) {
                        int idx = hash3D(n->m_seed, xi, yi, zi) & 255;

                        float vecX = xi - x + CELL_3D_x[idx];
                        float vecY = yi - y + CELL_3D_y[idx];
                        float vecZ = zi - z + CELL_3D_z[idx];

                        float newDistance = fabsf(vecX) + fabsf(vecY) + fabsf(vecZ);

                        if (newDistance < distance) {
                            distance = newDistance;
                            xc = xi;
                            yc = yi;
                            zc = zi;
                        }
                    }
                }
            }
            break;
        case Natural:
            for (int xi = xr - 1; xi <= xr + 1; xi++) {
                for (int yi = yr - 1; yi <= yr + 1; yi++) {
                    for (int zi = zr - 1; zi <= zr + 1; zi++) {
                        int idx = hash3D(n->m_seed, xi, yi, zi) & 255;

                        float vecX = xi - x + CELL_3D_x[idx];
                        float vecY = yi - y + CELL_3D_y[idx];
                        float vecZ = zi - z + CELL_3D_z[idx];

                        float newDistance = (fabsf(vecX) + fabsf(vecY) + fabsf(vecZ)) + (vecX * vecX + vecY * vecY + vecZ * vecZ);

                        if (newDistance < distance) {
                            distance = newDistance;
                            xc = xi;
                            yc = yi;
                            zc = zi;
                        }
                    }
                }
            }
            break;
    }

    switch (n->m_cellularReturnType) {
        case CellValue:
            return valCoord3D(0, xc, yc, zc);
        case Distance:
            return distance - 1;
        default:
            return 0;
    }
}

__device__ float singleCellular2Edge(FastNoise* n, float x, float y, float z) {
    int xr = max(min(fastRound(x), MAX_CELL_POS), MIN_CELL_POS);
    int yr = max(min(fastRound(y), MAX_CELL_POS), MIN_CELL_POS);
    int zr = max(min(fastRound(z), MAX_CELL_POS), MIN_CELL_POS);

    float distance = 999999;
    float distance2 = 999999;

    switch (n->m_cellularDistanceFunction) {
        case Euclidean:
            for (int xi = xr - 1; xi <= xr + 1; xi++) {
                for (int yi = yr - 1; yi <= yr + 1; yi++) {
                    for (int zi = zr - 1; zi <= zr + 1; zi++) {
                       int idx = hash3D(n->m_seed, xi, yi, zi) & 255;
                        float vecX = xi - x + CELL_3D_x[idx];
                        float vecY = yi - y + CELL_3D_y[idx];
                        float vecZ = zi - z + CELL_3D_z[idx];

                        float newDistance = vecX * vecX + vecY * vecY + vecZ * vecZ;

                        distance2 = fmaxf(fminf(distance2, newDistance), distance);
                        distance = fminf(distance, newDistance);
                    }
                }
            }
            break;
        case Manhattan:
            for (int xi = xr - 1; xi <= xr + 1; xi++) {
                for (int yi = yr - 1; yi <= yr + 1; yi++) {
                    for (int zi = zr - 1; zi <= zr + 1; zi++) {
                        int idx = hash3D(n->m_seed, xi, yi, zi) & 255;

                        float vecX = xi - x + CELL_3D_x[idx];
                        float vecY = yi - y + CELL_3D_y[idx];
                        float vecZ = zi - z + CELL_3D_z[idx];

                        float newDistance = fabsf(vecX) + fabsf(vecY) + fabsf(vecZ);

                        distance2 = fmaxf(fminf(distance2, newDistance), distance);
                        distance = fminf(distance, newDistance);
                    }
                }
            }
            break;
        case Natural:
            for (int xi = xr - 1; xi <= xr + 1; xi++) {
                for (int yi = yr - 1; yi <= yr + 1; yi++) {
                    for (int zi = zr - 1; zi <= zr + 1; zi++) {
                        int idx = hash3D(n->m_seed, xi, yi, zi) & 255;

                        float vecX = xi - x + CELL_3D_x[idx];
                        float vecY = yi - y + CELL_3D_y[idx];
                        float vecZ = zi - z + CELL_3D_z[idx];

                        float newDistance = (fabsf(vecX) + fabsf(vecY) + fabsf(vecZ)) + (vecX * vecX + vecY * vecY + vecZ * vecZ);

                        distance2 = fmaxf(fminf(distance2, newDistance), distance);
                        distance = fminf(distance, newDistance);
                    }
                }
            }
            break;
        default:
            break;
    }

    switch (n->m_cellularReturnType) {
        case Distance2:
            return distance2 - 1;
        case Distance2Add:
            return distance2 + distance - 1;
        case Distance2Sub:
            return distance2 - distance - 1;
        case Distance2Mul:
            return distance2 * distance - 1;
        case Distance2Div:
            return distance / distance2 - 1;
        default:
            return 0;
    }
}

__device__ float getCellular(FastNoise* n, float x, float y, float z) {
    x *= n->m_frequency;
    y *= n->m_frequency;
    z *= n->m_frequency;

    switch (n->m_cellularReturnType) {
        case CellValue:
        case Distance:
            return singleCellular(n, x, y, z);
        default:
            return singleCellular2Edge(n, x, y, z);
    }
}
#endif // ADD_FEATURE_CELLULAR_NOISE

// MAIN FUNCTION
__device__ float getNoise(FastNoise* n, float x, float y, float z) {
    x *= n->m_frequency;
    y *= n->m_frequency;
    z *= n->m_frequency;

    switch (n->m_noiseType) {
        case Value:
#ifdef ADD_FEATURE_VALUE_NOISE
            return singleValue(n, n->m_seed, x, y, z);
#else
            return 0.f;
#endif
        case ValueFractal:
#ifdef ADD_FEATURE_VALUE_NOISE
            switch (n->m_fractalType) {
                case FBM:
                    return singleValueFractalFBM(n, x, y, z);
                case Billow:
                    return singleValueFractalBillow(n, x, y, z);
                case RigidMulti:
                    return singleValueFractalRigidMulti(n, x, y, z);
                default:
                    return 0;
            }
#else
            return 0.f;
#endif
        case Perlin:
#ifdef ADD_FEATURE_PERLIN_NOISE
            return singlePerlin(n, n->m_seed, x, y, z);
#else
            return 0.f;
#endif
        case PerlinFractal:
#ifdef ADD_FEATURE_PERLIN_NOISE
            switch (n->m_fractalType) {
                case FBM:
                    return singlePerlinFractalFBM(n, x, y, z);
                case Billow:
                    return singlePerlinFractalBillow(n, x, y, z);
                case RigidMulti:
                    return singlePerlinFractalRigidMulti(n, x, y, z);
                default:
                    return 0;
            }
#else
            return 0.f;
#endif
        case Simplex:
#ifdef ADD_FEATURE_SIMPLEX_NOISE
            return singleSimplex(n->m_seed, x, y, z);
#else
            return 0.f;
#endif
        case SimplexFractal:
#ifdef ADD_FEATURE_SIMPLEX_NOISE
            switch (n->m_fractalType) {
                case FBM:
                    return singleSimplexFractalFBM(n, x, y, z);
                case Billow:
                    return singleSimplexFractalBillow(n, x, y, z);
                case RigidMulti:
                    return singleSimplexFractalRigidMulti(n, x, y, z);
                default:
                    return 0;
            }
#else
            return 0.f;
#endif
        case Cellular:
#ifdef ADD_FEATURE_CELLULAR_NOISE
            switch (n->m_cellularReturnType) {
                case CellValue:
                case Distance:
                    return singleCellular(n, x, y, z);
                default:
                    return singleCellular2Edge(n, x, y, z);
            }
#else
            return 0.f;
#endif
        case WhiteNoise:
#ifdef ADD_FEATURE_WHITE_NOISE
            return getWhiteNoise(n, x, y, z);
#else
            return 0.f;
#endif
        case Cubic:
#ifdef ADD_FEATURE_CUBIC_NOISE
            return singleCubic(n, n->m_seed, x, y, z);
#else
            return 0.f;
#endif
        case CubicFractal:
#ifdef ADD_FEATURE_CUBIC_NOISE
            switch (n->m_fractalType) {
                case FBM:
                    return singleCubicFractalFBM(n, x, y, z);
                case Billow:
                    return singleCubicFractalBillow(n, x, y, z);
                case RigidMulti:
                    return singleCubicFractalRigidMulti(n, x, y, z);
                default:
                    return 0;
            }
#else
            return 0.f;
#endif
        default:
            return 0;
    }
}
#endif // ADD_FEATURE_FAST_NOISE

//------------- START of JS CODE--------------------------
// vector operations 2D,3D, 4D

#define vec2 float2
#define vec3 float3
#define vec4 float4

 __device__ float2 operator+(const float2 &a, float d) {
   return make_float2(a.x+d, a.y+d);
 }

 __device__ float2 operator-(const float2 &a, float d) {
   return make_float2(a.x-d, a.y-d);
 }
  __device__ float2 operator*(const float2 &a, float d) {
   return make_float2(a.x*d, a.y*d);
 }
  __device__ float2 operator/(const float2 &a, float d) {
   return make_float2(a.x/d, a.y/d);
 }
 __device__ float2 operator+(const float2 &a, const float2 &b) {
   return make_float2(a.x+b.x, a.y+b.y);
 }
 __device__ float2 operator-(const float2 &a, const float2 &b) {
   return make_float2(a.x-b.x, a.y-b.y);
 }
 __device__ float2 operator*(const float2 &a, const float2 &b) {
   return make_float2(a.x*b.x, a.y*b.y);
 }
  __device__ float2 operator/(const float2 &a, const float2 &b) {
   return make_float2(a.x/b.x, a.y/b.y);
 }
 
  __device__ float3 operator+(const float3 &a, float d) {
   return make_float3(a.x+d, a.y+d, a.z+d);
 }
  __device__ float3 operator-(const float3 &a, float d) {
   return make_float3(a.x-d, a.y-d, a.z-d);
 }
  __device__ float3 operator*(const float3 &a, float d) {
   return make_float3(a.x*d, a.y*d, a.z*d);
 }
  __device__ float3 operator/(const float3 &a, float d) {
   return make_float3(a.x/d, a.y/d, a.z/d);
 }
  
__device__ float3 operator+(const float3 &a, const float3 &b) {
   return make_float3(a.x+b.x, a.y+b.y, a.z+b.z  );
 } 
 __device__ float3 operator-(const float3 &a, const float3 &b) {
   return make_float3(a.x-b.x, a.y-b.y, a.z-b.z  );
 }
  __device__ float3 operator*(const float3 &a, const float3 &b) {
   return make_float3(a.x*b.x, a.y*b.y, a.z*b.z  );
 }
  __device__ float3 operator/(const float3 &a, const float3 &b) {
   return make_float3(a.x/b.x, a.y/b.y, a.z/b.z  );
 }
 
   __device__ float4 operator+(const float4 &a, float d) {
   return make_float4(a.x+d, a.y+d, a.z+d, a.w+d);
 }
   __device__ float4 operator-(const float4 &a, float d) {
   return make_float4(a.x-d, a.y-d, a.z-d, a.w-d);
 }
   __device__ float4 operator*(const float4 &a, float d) {
   return make_float4(a.x*d, a.y*d, a.z*d, a.w*d);
 }
   __device__ float4 operator/(const float4 &a, float d) {
   return make_float4(a.x/d, a.y/d, a.z/d, a.w/d);
 }
 
 __device__ float4 operator+(const float4 &a, const float4 &b) {
   return make_float4(a.x+b.x, a.y+b.y, a.z+b.z, a.w+b.w  );
 } 
 __device__ float4 operator-(const float4 &a, const float4 &b) {
   return make_float4(a.x-b.x, a.y-b.y, a.z-b.z, a.w-b.w  );
 } 
  __device__ float4 operator*(const float4 &a, const float4 &b) {
   return make_float4(a.x*b.x, a.y*b.y, a.z*b.z, a.w*b.w  );
 }
 __device__ float4 operator/(const float4 &a, const float4 &b) {
   return make_float4(a.x/b.x, a.y/b.y, a.z/b.z, a.w/b.w  );
 } 

__device__ float atan(float n, float d)
{
	return atanf(n/d);
}

__device__ float atan2 (float y, float x)
{
	return atan2f(y,x);
}
__device__ float sqrt(float a)
{
	return sqrtf(a);
}

__device__ float2 sqrt(float2 a)
{
	return make_float2(sqrtf(a.x),sqrtf(a.y));
}

__device__ float3 sqrt(float3 a)
{
	return make_float3( sqrtf(a.x),sqrtf(a.y),sqrtf(a.z) );
}
		
__device__ float4 sqrt(float4 a)
{
	return make_float4( sqrtf(a.x),sqrtf(a.y),sqrtf(a.z),sqrtf(a.w) );
}	

__device__ float pow(float x,float y)
{
	return  powf(x,y);
}
		
__device__ float2 pow(float2 x,float2 y)
{
	float xr= powf(x.x,y.x);
	float yr= powf(x.y,y.y);
	return make_float2(xr,yr);
}

__device__ float3 pow(float3 x,float3 y)
{
	float xr= powf(x.x,y.x);
	float yr= powf(x.y,y.y);
	float zr= powf(x.z,y.z);
	return make_float3(xr,yr,zr);
}
		
__device__ float4 pow(float4 x,float4 y)
{
	float xr= powf(x.x,y.x);
	float yr= powf(x.y,y.y);
	float zr= powf(x.z,y.z);
	float wr= powf(x.w,y.w);
	return make_float4(xr,yr,zr,wr);
}

		
__device__ float2 expf(float2 x)
{
	float xr= expf(x.x);
	float yr= expf(x.y);
	return make_float2(xr,yr);
}

__device__ float3 expf(float3 x)
{
	float xr= expf(x.x);
	float yr= expf(x.y);
	float zr= expf(x.z);
	return make_float3(xr,yr,zr);
}
		
__device__ float4 expf(float4 x)
{
	float xr= expf(x.x);
	float yr= expf(x.y);
	float zr= expf(x.z);
	float wr= expf(x.w);
	return make_float4(xr,yr,zr,wr);
}
		

__device__ float2 exp2f(float2 x)
{
 	float xr= powf(2.0,x.x);
 	float yr= powf(2.0,x.y);
 	return make_float2(xr,yr);
}

__device__ float3 exp2f(float3 x)
{
 	float xr= powf(2.0,x.x);
 	float yr= powf(2.0,x.y);
 	float zr= powf(2.0,x.z);
 	return make_float3(xr,yr,zr);
}

__device__ float4 exp2f(float4 x)
{
 	float xr= powf(2.0,x.x);
 	float yr= powf(2.0,x.y);
 	float zr= powf(2.0,x.z);
 	float wr= powf(2.0,x.w);
 	return make_float4(xr,yr,zr,wr);
}

__device__ float log2(float d) {
	return log2f(d);
}

__device__ float sin(float x)
{
    return sinf(x);
}

__device__ float2 sinf(float2 x)
{
	float v1,v2;
	v1=sinf(x.x);
	v2=sinf(x.y);
	return make_float2(v1,v2);
}

__device__ float2 sin(float2 x)
{
	float v1,v2;
	v1=sinf(x.x);
	v2=sinf(x.y);
	return make_float2(v1,v2);
}

__device__ float3 sinf(float3 a)
{
	float v1,v2,v3;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	return make_float3(v1,v2,v3);
}

__device__ float3 sin(float3 a)
{
	float v1,v2,v3;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	return make_float3(v1,v2,v3);
}

__device__ float4 sin(float4 a)
{
	float v1,v2,v3,v4;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	v4=sinf(a.w);
	return make_float4(v1,v2,v3,v4);
}

__device__ float4 sinf(float4 a)
{
	float v1,v2,v3,v4;
	v1=sinf(a.x);
	v2=sinf(a.y);
	v3=sinf(a.z);
	v4=sinf(a.w);
	return make_float4(v1,v2,v3,v4);
}
		
__device__ float cos(float x)
{
	return cosf(x);
}

__device__ float2 cosf(float2 x)
{
	float v1,v2;
	v1=cosf(x.x);
	v2=cosf(x.y);
	return make_float2(v1,v2);
}

__device__ float2 cos(float2 x)
{
	float v1,v2;
	v1=cosf(x.x);
	v2=cosf(x.y);
	return make_float2(v1,v2);
}

__device__ float3 cosf(float3 a)
{
	float v1,v2,v3;
	v1=cosf(a.x);
	v2=cosf(a.y);
	v3=cosf(a.z);
	return make_float3(v1,v2,v3);
}

__device__ float3 cos(float3 a)
{
	float v1,v2,v3;
	v1=cosf(a.x);
	v2=cosf(a.y);
	v3=cosf(a.z);
	return make_float3(v1,v2,v3);
}

__device__ float4 cos(float4 a)
{
	float v1,v2,v3,v4;
	v1=cosf(a.x);
	v2=cosf(a.y);
	v3=cosf(a.z);
	v4=cosf(a.w);
	return make_float4(v1,v2,v3,v4);
}

		
__device__ float length(float2 a)
{
	return sqrtf(a.x*a.x+a.y*a.y);
}

__device__ float length(float3 a)
{
	return sqrtf(a.x*a.x+a.y*a.y+a.z*a.z);
}

__device__ float length(float4 a)
{
	return  sqrtf(a.x*a.x+a.y*a.y+a.z*a.z+ a.w*a.w);
}

__device__ float2 abs(float2 v)
{
	return make_float2(fabsf(v.x),fabsf(v.y));
}

__device__ float3 abs(float3 v)
{
	return make_float3(fabsf(v.x),fabsf(v.y),fabsf(v.z));
}

__device__ float4 abs(float4 v)
{
	return make_float4(fabsf(v.x),fabsf(v.y),fabsf(v.z),fabsf(v.w));
}

__device__ float sign(float v)
{
    float val=0.0;
    if(v>0.0)
	   val=1.0;
	else if(v<0.0)
	   val=-1.0;
	else
	   val=0.0;
	return val;
}

__device__ float2 sign(float2 v)
{
	return make_float2(sign(v.x),sign(v.y));
}
	
	__device__ float3 sign(float3 v)
{
	return make_float3(sign(v.x),sign(v.y),sign(v.z));
}

__device__ float dot(float v1,float v2)
{
	return v1*v2;
}

__device__ float dot(float2 v1,float2 v2)
{
	return v1.x*v2.x + v1.y*v2.y;
}

__device__ float dot(float3 v1,float3 v2)
{
	return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z;
}

__device__ float dot(float4 v1,float4 v2)
{
	return v1.x*v2.x + v1.y*v2.y + v1.z*v2.z + v1.w*v2.w;
}
__device__ float3 cross(float3 x,float3 y)
{
	float x1,y1,z1;
	x1=x.y*y.z -y.y*x.z;
	y1=x.z*y.x-y.z*x.x;
	z1=x.x*y.y-y.x*x.y;
	return make_float3(x1,y1,z1);
}

__device__ float mod(float x,float y)
{
	return x- y*floorf(x/y);
}

__device__ float2 mod(float2 x,float y)
{
	float x1,y1;
	x1=	x.x- y*floorf(x.x/y);
	y1=	x.y- y*floorf(x.y/y);
	return make_float2(x1,y1);
}

__device__ float2 mod(float2 x,float2 y)
{

	float x1,y1;
	x1=	x.x- y.x*floorf(x.x/y.x);
	y1=	x.y- y.y*floorf(x.y/y.y);
	return make_float2(x1,y1);
}

__device__ float3 mod(float3 x,float y)
{
	float x1,y1,z1;
	x1=	x.x- y*floorf(x.x/y);
	y1=	x.y- y*floorf(x.y/y);
	z1=	x.z- y*floorf(x.z/y);
	return make_float3(x1,y1,z1);
}

__device__ float3 mod(float3 x,float3 y)
{
	float x1,y1,z1;
	x1 = 	x.x- y.x*floorf(x.x/y.x);
	y1 = 	x.y- y.y*floorf(x.y/y.y);
	z1 = 	x.z- y.z*floorf(x.z/y.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 mod(float4 x,float y)
{
	float x1,y1,z1,w1;
	x1=	x.x- y*floorf(x.x/y);
	y1=	x.y- y*floorf(x.y/y);
	z1=	x.z- y*floorf(x.z/y);
	w1=	x.w- y*floorf(x.w/y);
	return make_float4(x1,y1,z1,w1);
}
__device__ float step(float lim, float x)
{
    return (x<lim)?0.0f:1.0f;
}

__device__ float2 step(float  lim, float2 x)
{
	float x1,y1;
	x1=(x.x<lim)?0.0f:1.0f;
	y1=(x.y<lim)?0.0f:1.0;
	return make_float2(x1,y1);
}
		
__device__ float2 step(float2 lim, float2 x)
{
	float x1,y1;
	x1=(x.x<lim.x)?0.0f:1.0f;
	y1=(x.y<lim.y)?0.0f:1.0f;
	return make_float2(x1,y1);
}

__device__ float3 step(float lim, float3 x)
{
	float x1,y1,z1;
	x1=(x.x<lim)?0.0f:1.0f;
	y1=(x.y<lim)?0.0f:1.0f;
	z1=(x.z<lim)?0.0f:1.0f;
	return make_float3(x1,y1,z1);
}
		
__device__ float3 step(float3 lim, float3 x)
{
	float x1,y1,z1;
	x1=(x.x<lim.x)?0.0f:1.0f;
	y1=(x.y<lim.y)?0.0f:1.0f;
	z1=(x.z<lim.z)?0.0f:1.0f;
	return make_float3(x1,y1,z1);
}

__device__ float4 step(float4 lim, float4 x)
{
	float x1,y1,z1,w1;
	x1=(x.x<lim.x)?0.0f:1.0f;
	y1=(x.y<lim.y)?0.0f:1.0f;
	z1=(x.z<lim.z)?0.0f:1.0f;
	w1=(x.w<lim.w)?0.0f:1.0f;
	return make_float4(x1,y1,z1,w1);
}

__device__ float ceil(float x)
{
   return ceilf(x);
}

__device__ float2 ceil(float2 v)
{
   return make_float2(ceilf(v.x),ceilf(v.y));
}

__device__ float3 ceil(float3 v)
{
   return make_float3(ceilf(v.x),ceilf(v.y),ceilf(v.z));
}

__device__ float4 ceil(float4 v)
{
   return make_float4(ceilf(v.x),ceilf(v.y),ceilf(v.z),ceilf(v.w));
}

__device__ float2 floorf(float2 v)
{
	return make_float2(floorf(v.x),floorf(v.y));
}

__device__ float3 floorf(float3 v)
{
	return make_float3(floorf(v.x),floorf(v.y),floorf(v.z));
}

__device__ float4 floorf(float4 v)
{
	return make_float4(floorf(v.x),floorf(v.y),floorf(v.z),floorf(v.w));
}

	
		
__device__ float2 truncf(float2 v)
{
	float x1,y1;
	x1=truncf(v.x);
	y1=truncf(v.y);
	return make_float2(x1,y1);
}

__device__ float3 truncf(float3 v)
{
	float x1,y1,z1;
	x1=truncf(v.x);
	y1=truncf(v.y);
	z1=truncf(v.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 truncf(float4 v)
{
	float x1,y1,z1,w1;
	x1=truncf(v.x);
	y1=truncf(v.y);
	z1=truncf(v.z);
	w1=truncf(v.w);
	return make_float4(x1,y1,z1,w1);
}

		
__device__ float2 roundf(float2 v)
{
	float x1,y1;
	x1=roundf(v.x);
	y1=roundf(v.y);
	return make_float2(x1,y1);
}

__device__ float3 roundf(float3 v)
{
	float x1,y1,z1;
	x1=roundf(v.x);
	y1=roundf(v.y);
	z1=roundf(v.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 roundf(float4 v)
{
	float x1,y1,z1,w1;
	x1=roundf(v.x);
	y1=roundf(v.y);
	z1=roundf(v.z);
	w1=roundf(v.w);
	return make_float4(x1,y1,z1,w1);
}
	
		
__device__ float2 ceilf(float2 v)
{
	float x1,y1;
	x1=ceilf(v.x);
	y1=ceilf(v.y);
	return make_float2(x1,y1);
}

__device__ float3 ceilf(float3 v)
{
	float x1,y1,z1;
	x1=ceilf(v.x);
	y1=ceilf(v.y);
	z1=ceilf(v.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 ceilf(float4 v)
{
	float x1,y1,z1,w1;
	x1=ceilf(v.x);
	y1=ceilf(v.y);
	z1=ceilf(v.z);
	w1=ceilf(v.w);
	return make_float4(x1,y1,z1,w1);
}	
__device__ float fract(float x)
{
	return x-floorf(x);
}

__device__ float2 fract(float2 x)
{
	return make_float2(x.x-floorf(x.x),x.y-floorf(x.y));
}

__device__ float3 fract(float3 x)
{
	float xr,yr,zr;
	xr=x.x-floorf(x.x);
	yr=x.y-floorf(x.y);
	zr=x.z-floorf(x.z);
	return make_float3(xr,yr,zr);
}

__device__ float4 fract(float4 x)
{
	float xr,yr,zr,wr;
	xr=x.x-floorf(x.x);
	yr=x.y-floorf(x.y);
	zr=x.z-floorf(x.z);
	wr=x.w-floorf(x.w);
	return make_float4(xr,yr,zr,wr);
}

__device__ float mix(float x,float y, float a)
{
	float z;
	z= (x*(1.0f-a) + y*a);
	return z;
}

__device__ float2 mix(float2 x,float2 y, float a)
{
	float x1,y1;
	x1= (x.x*(1.0f-a) + y.x*a);
	y1= (x.y*(1.0f-a) + y.y*a);
	return make_float2(x1,y1);
}  

__device__ float3 mix(float3 x,float3 y, float a)
{
	return make_float3((x.x*(1.0f-a) + y.x*a), (x.y*(1.0f-a) + y.y*a), (x.z*(1.0f-a) + y.z*a));
}  

__device__ float4 mix(float4 x,float4 y, float a)
{
	float vx,vy,vz,vw;
	vx= (x.x*(1.0f-a) + y.x*a);
	vy= (x.y*(1.0f-a) + y.y*a);
	vz= (x.z*(1.0f-a) + y.z*a);
	vw= (x.w*(1.0f-a) + y.w*a);
	return make_float4(vx,vy,vz,vw);
} 

__device__ float clamp(float val, float min, float  max) {
    return fmaxf(min, fminf(max, val));
}

__device__ float2 clamp(float2 x,float minVal, float maxVal)
{
	float vx,vy;
	vx=fminf(fmaxf(x.x, minVal), maxVal);
	vy=fminf(fmaxf(x.y, minVal), maxVal);
	return make_float2(vx,vy);
}

__device__ float3 clamp(float3 x,float minVal, float maxVal)
{
	float vx,vy,vz;
	vx=fminf(fmaxf(x.x, minVal), maxVal);
	vy=fminf(fmaxf(x.y, minVal), maxVal);
	vz=fminf(fmaxf(x.z, minVal), maxVal);
	return make_float3(vx,vy,vz);
}

__device__ float4 clamp(float4 x,float minVal, float maxVal)
{
	float vx,vy,vz,vw;
	vx=fminf(fmaxf(x.x, minVal), maxVal);
	vy=fminf(fmaxf(x.y, minVal), maxVal);
	vz=fminf(fmaxf(x.z, minVal), maxVal);
	vw=fminf(fmaxf(x.w, minVal), maxVal);
	return make_float4(vx,vy,vz,vw);
}

__device__ float smootherstep(float edge0, float edge1, float x) {
    x = clamp((x - edge0) / (edge1 - edge0), 0.0f, 1.0f);
    return x * x * x * (x * (x * 6.f - 15.f) + 10.f);
 }
  

__device__ float smoothstep(float edge0, float edge1, float x)
{
	float t= clamp((x - edge0) / (edge1 - edge0), 0.0f, 1.0f);
	return t * t * (3.0f - 2.0f * t);
}

__device__ float2 smoothstep(float e0,float e1,float2 x)
{
	return make_float2(smoothstep(e0,e1,x.x),smoothstep(e0,e1,x.y));
}
		
		
__device__ float2 smoothstep(float2 e0,float2 e1, float2 x) 
{

	return make_float2(smoothstep(e0.x,e1.x,x.x),smoothstep(e0.y,e1.y,x.y));
}
__device__ float3 smoothstep(float e0,float  e1, float3 x) 
{
	return make_float3(smoothstep(e0,e1,x.x),smoothstep(e0,e1,x.y),smoothstep(e0,e1,x.z));
}
		
__device__ float3 smoothstep(float3 e0,float3 e1, float3 x) 
{
	return make_float3(smoothstep(e0.x,e1.x,x.x),smoothstep(e0.y,e1.y,x.y),smoothstep(e0.z,e1.z,x.z));
}

__device__ float4 smoothstep(float e0,float e1, float4 x) 
{
	return make_float4(smoothstep(e0,e1,x.x),smoothstep(e0,e1,x.y),smoothstep(e0,e1,x.z),smoothstep(e0,e1,x.w));
}
		
__device__ float4 smoothstep(float4 e0,float4 e1, float4 x) 
{
	return make_float4(smoothstep(e0.x,e1.x,x.x),smoothstep(e0.y,e1.y,x.y),smoothstep(e0.z,e1.z,x.z),smoothstep(e0.w,e1.w,x.w));
}

__device__ float distance(float2 v1,float2 v2)
{
		float2 dif=v1-v2;
		return sqrt(dif.x*dif.x+dif.y*dif.y);
}

__device__ float distance(float3 v1,float3 v2)
{
	float3 dif=v1-v2;
	return sqrt(dif.x*dif.x+dif.y*dif.y + dif.z*dif.z);
}

__device__ float distance(float4 v1,float4 v2)
{
	float4 dif=v1-v2;
	return sqrt(dif.x*dif.x+dif.y*dif.y + dif.z*dif.z + dif.w*dif.w);
}

__device__ float2 normalize(float2 v1)
{
	float x= v1.x/length(v1);
	float y= v1.y/length(v1);
	return make_float2(x,y);
}

__device__ float3 normalize(float3 v1)
{
	float x= v1.x/length(v1);
	float y= v1.y/length(v1);
	float z= v1.z/length(v1);
	return make_float3(x,y,z);
}
		
__device__ float4 normalize(float4 v1)
{
	float x= v1.x/length(v1);
	float y= v1.y/length(v1);
	float z= v1.z/length(v1);
	float w= v1.w/length(v1);
	return make_float4(x,y,z,w);
}


__device__ float2 min(float2 x, float2 y)
{
	float x1,y1;
	x1=fminf(x.x,y.x);
	y1=fminf(x.y,y.y);
	return make_float2(x1,y1);
}
		
__device__ float3 min(float3 x, float3 y)
{
	float x1,y1,z1;
	x1=fminf(x.x,y.x);
	y1=fminf(x.y,y.y);
	z1=fminf(x.z,y.z);
	return make_float3(x1,y1,z1);
}

__device__ float4 min(float4 x, float4 y)
{
	float x1,y1,z1,w1;
	x1=fminf(x.x,y.x);
	y1=fminf(x.y,y.y);
	z1=fminf(x.z,y.z);
	w1=fminf(x.w,y.w);
	return make_float4(x1,y1,z1,w1);
}
		
__device__ float2 min(float2 x, float y)
{
	float x1,y1;
	x1=fminf(x.x,y);
	y1=fminf(x.y,y);
	return make_float2(x1,y1);
}
		
__device__ float3 min(float3 x, float y)
{
	float x1,y1,z1;
	x1=fminf(x.x,y);
	y1=fminf(x.y,y);
	z1=fminf(x.z,y);
	return make_float3(x1,y1,z1);
}

__device__ float4 min(float4 x, float y)
{
	float x1,y1,z1,w1;
	x1=fminf(x.x,y);
	y1=fminf(x.y,y);
	z1=fminf(x.z,y);
	w1=fminf(x.w,y);
	return make_float4(x1,y1,z1,w1);
}


__device__ float2 max(float2 x,float2 y)
{
	float v1,v2;
	v1=fmaxf(x.x,y.x);
	v2=fmaxf(x.y,y.y);
	return make_float2(v1,v2);
}

__device__ float3 max(float3 x,float3 y)
{
	float v1,v2,v3;
	v1=fmaxf(x.x,y.x); 
	v2=fmaxf(x.y,y.y);
	v3=fmaxf(x.z,y.z);
	return make_float3(v1,v2,v3);
}

__device__ float4 max(float4 x,float4 y)
{
	float v1,v2,v3,v4;
	v1=fmaxf(x.x,y.x);
	v2=fmaxf(x.y,y.y);
	v3=fmaxf(x.z,y.z);
	v4=fmaxf(x.w,y.w);
	return make_float4(v1,v2,v3,v4);
}

__device__ float2 fmaxf(float2 x,float y)
{
	float v1,v2;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	return make_float2(v1,v2);
}
__device__ float2 max(float2 x,float y)
{
	float v1,v2;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	return make_float2(v1,v2);
}

__device__ float3 max(float3 x,float y)
{
	float v1,v2,v3;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	v3=fmaxf(x.z,y);
	return make_float3(v1,v2,v3);
}
	
__device__ float4 max(float4 x,float y)
{
	float v1,v2,v3,v4;
	v1=fmaxf(x.x,y);
	v2=fmaxf(x.y,y);
	v3=fmaxf(x.z,y);
	v4=fmaxf(x.w,y);
	return make_float4(v1,v2,v3,v4);
}


struct __align__(8) Mat2 {
	 float a00;
	 float a01;
	 float a10;
	 float a11;
};

__device__ void Mat2_Init(Mat2 *m, float v00, float v10, float v01, float v11) {
  m->a00 = v00;
  m->a01 = v01;
  m->a10 = v10;
  m->a11 = v11;
}

__device__ void Mat2_Init(Mat2 *m, float4 v) {
  m->a00 = v.x;
  m->a10 = v.y;
  m->a01 = v.z;
  m->a11 = v.w;
}

__device__ void Mat2_Init(Mat2 *m, float2 v1, float2 v2) {
  m->a00 = v1.x;
  m->a10 = v1.y;
  m->a01 = v2.x;
  m->a11 = v2.y;
}

__device__ float2 times(Mat2 *m, float2 v){
   	return  make_float2(m->a00*v.x + m->a01*v.y , m->a10*v.x + m->a11*v.y); 
}

__device__ void add(Mat2 *m, float v) {
  m->a00 += v;
  m->a10 += v;
  m->a01 += v;
  m->a11 += v;
}

__device__ void minus(Mat2 *m, float v) {
  m->a00 -= v;
  m->a10 -= v;
  m->a01 -= v;
  m->a11 -= v;
}

__device__ void times(Mat2 *m, float v) {
  m->a00 *= v;
  m->a10 *= v;
  m->a01 *= v;
  m->a11 *= v;
}

__device__ void division(Mat2 *m, float v) {
  m->a00 /= v;
  m->a10 /= v;
  m->a01 /= v;
  m->a11 /= v;
}

struct __align__(8) Mat3 {
	 float a00;
	 float a10;
	 float a20;
	 float a01;
	 float a11;
	 float a21;
	 float a02;
	 float a12;
	 float a22;
};

__device__ void Mat3_Init(Mat3 *m, float v00, float v10, float v20, float v01, float v11, float v21, float v02, float v12, float v22 ) {
  m->a00 = v00;
  m->a10 = v10;
  m->a20 = v20;
  m->a01 = v01;
  m->a11 = v11;
  m->a21 = v21;
  m->a02 = v02;
  m->a12 = v12;
  m->a22 = v22;
}

__device__ void Mat3_Init(Mat3 *m, float3 v1, float3 v2, float3 v3) {
  m->a00 = v1.x;
  m->a10 = v1.y;
  m->a20 = v1.z;
  m->a01 = v2.x;
  m->a11 = v2.y;
  m->a21 = v2.z;
  m->a02 = v3.x;
  m->a12 = v3.y;
  m->a22 = v3.z;
}

__device__ float3 times(Mat3 *m, float3 v){
     return make_float3(m->a00*v.x + m->a01*v.y + m->a02*v.z , m->a10*v.x + m->a11*v.y + m->a12*v.z , m->a20*v.x + m->a21*v.y + m->a22*v.z);
   	
}

__device__ void add(Mat3 *m, float v) {
  m->a00 += v;
  m->a10 += v;
  m->a20 += v;
  m->a01 += v;
  m->a11 += v;
  m->a21 += v;
  m->a02 += v;
  m->a12 += v;
  m->a22 += v;
}

__device__ void minus(Mat3 *m, float v) {
  m->a00 -= v;
  m->a10 -= v;
  m->a20 -= v;
  m->a01 -= v;
  m->a11 -= v;
  m->a21 -= v;
  m->a02 -= v;
  m->a12 -= v;
  m->a22 -= v;
}

__device__ void times(Mat3 *m, float v) {
  m->a00 *= v;
  m->a10 *= v;
  m->a20 *= v;
  m->a01 *= v;
  m->a11 *= v;
  m->a21 *= v;
  m->a02 *= v;
  m->a12 *= v;
  m->a22 *= v;
}

__device__ void division(Mat3 *m, float v) {
  m->a00 /= v;
  m->a10 /= v;
  m->a20 /= v;
  m->a01 /= v;
  m->a11 /= v;
  m->a21 /= v;
  m->a02 /= v;
  m->a12 /= v;
  m->a22 /= v;
}

struct __align__(8) Mathc {
	 float a;
	 float b;
	 float c;
	 float d;
	 float e;
	 float f;	 
};

__device__ Mat3 rotEuler (float3 s) {
		float 	sa = sinf(s.x),
		ca = cosf(s.x),
		sb = sinf(s.y),
		cb = cosf(s.y),
		sc = sinf(s.z),
		cc = cosf(s.z);
		Mat3 M;
		Mat3_Init (&M,make_float3(cb*cc, -cb*sc, sb),
		              make_float3(sa*sb*cc+ca*sc, -sa*sb*sc+ca*cc, -sa*cb),
		              make_float3(-ca*sb*cc+sa*sc, ca*sb*sc+sa*cc, ca*cb)  );
	return M;
}

__device__ float distance_color(float p_red,float p_green,float p_blue,float red,float green,float blue)
{
	float dist_r = fabsf(p_red - red);
	float dist_g = fabsf(p_green - green);
	float dist_b = fabsf(p_blue - blue);
	float dist_3d_sqd = (dist_r * dist_r) + (dist_g * dist_g) + (dist_b * dist_b);
	return dist_3d_sqd;
}

__device__ float2  transfhcf (float2 xy,float a,float b,float c,float d,float e,float f)
{
  float xt=a*xy.x+b*xy.y+c;
  float yt=d*xy.x+e*xy.y+f;
  return make_float2(xt,yt);
}

__device__ float greyscale(int r,int  g,int b)
{
  int lum,red,green,blue;
  red = (r * 0.299);         
  green = (g * 0.587);         
  blue = (b * 0.114);    
  lum = red + green + blue;    
  return (float)lum/255.0f;
}

__device__ int3 dbl2int(float3 theColor)
  	{
  		int red   =  max(0, min(255, (int)floorf(theColor.x * 256.0f)));
  		int green =  max(0, min(255, (int)floorf(theColor.y * 256.0f)));
  		int blue  =  max(0, min(255, (int)floorf(theColor.z * 256.0f)));
  		return make_int3(red,green,blue);
  	}
	
	
__device__ float3  hsv2rgb (float3 c) 
	{
	  float4 K = make_float4(1.0f, 2.0f / 3.0f, 1.0f / 3.0f, 3.0f);
	  float3 p = abs(fract(make_float3(c.x,c.x,c.x)+(make_float3(K.x,K.y,K.z)))*(6.0f)-(make_float3(K.w,K.w,K.w)));
	  return mix(make_float3(K.x,K.x,K.x), clamp(p - make_float3(K.x,K.x,K.x), 0.0f, 1.0f), c.y)*c.z;
	}
	
	struct __align__(8) Jacobi_elliptic_result
{ float cn;
  float dn;
  float sn;
};
	
__device__ void Jacobi_elliptic( float uu, float emmc, Jacobi_elliptic_result *res)
{
    res->cn=0.0;
	res->dn=0.0;
	res->sn=0.0;
    
    float CA = 0.0003; 
    float a, b, c = 0.0, d = 0.0, em[13] , en[13];
    int bo;
    int l = 0;
    int ii;
    int i;
    
    
    float emc = emmc;
    float u = uu;
    if (emc != 0.0) {
      bo = 0;
      if (emc < 0.0)
        bo = 1;
      if (bo != 0) {
        d = 1.0 - emc;
        emc = -emc / d;
        d = sqrtf(d);
        u = d * u;
      }
      a = 1.0;
      res->dn = 1.0;
      
      for (i = 0; i < 8; i++) {
        l = i;
        em[i] = a;
        emc = sqrtf(emc);
        en[i] = emc;
        c = 0.5 * (a + emc);
        if (fabsf(a - emc) <= CA * a)
          break;
        emc = a * emc;
        a = c;
      }
      u = c * u;
      res->sn = sinf(u);
      res->cn = cosf(u);
      if (res->sn != 0.0) {
        a = res->cn / res->sn;
        c = a * c;
        for (ii = l; ii >= 0; --ii) {
          b = em[ii];
          a = c * a;
          c = res->dn * c;
          res->dn = (en[ii] + a) / (b + a);
          a = c / b;
        }
        a = 1.0 / sqrtf(c * c + 1.0);
        if (res->sn < 0.0)
          (res->sn) = -a;
        else
          res->sn = a;
        res->cn = c * (res->sn);
      }
      if (bo != 0) {
        a = res->dn;
        res->dn = res->cn;
        res->cn = a;
        res->sn = (res->sn) / d;
      }
    } else {
      res->cn = 1.0 / coshf(u);
      res->dn = res->cn;
      (res->sn) = tanhf(u);
    }
}

//------------- END of JS CODE--------------------------


struct __align__(8) Complex
{
  float per_fix;
  float re;
  float im;
  float save_re;
  float save_im;
};

__device__ void Complex_Init(Complex *c, float Rp, float Ip) {
  c->re = Rp;
  c->im = Ip;
  c->save_re = 0.f;
  c->save_im = 0.f;
  c->per_fix = 0.f;  
}


	
__device__ float Complex_Mag2(Complex *c) {
    return c->re * c->re + c->im * c->im;
}
  
__device__ float Complex_MagInv(Complex *c) {
    float M2 = Complex_Mag2(c);
    return (M2 < 1e-10 ? 1.0f : 1.0f / M2);
}
  
__device__ void Complex_Recip(Complex *c) {
    float mi = Complex_MagInv(c);
    c->re = c->re * mi;
    c->im = -c->im * mi;
}

__device__ void Complex_Dec(Complex *c) {
  c->re -= 1.0f;
}

__device__ void Complex_Inc(Complex *c) {
  c->re += 1.0f;
}

__device__ void Complex_Neg(Complex *c) {
  c->re = -c->re;
  c->im = -c->im;
}
  
__device__ void Complex_Div(Complex *c, Complex *zz) {
  float r2 = c->im * zz->im + c->re * zz->re;
  float i2 = c->im * zz->re - c->re * zz->im;
  float M2 = Complex_MagInv(zz);
  c->re = r2 * M2;
  c->im = i2 * M2;
}
  
  __device__ void Complex_DivR(Complex *T,Complex *zz) {
	float r2 = zz->im * T->im + zz->re * T->re;
	float i2 = zz->im * T->re - zz->re * T->im;
	float M2 = Complex_MagInv(T);
	T->re = r2 * M2;
	T->im = i2 * M2;
} 

__device__ void Complex_Copy(Complex *c, Complex *zz) {
  c->re = zz->re;
  c->im = zz->im;
}
  
__device__ float Complex_Mag2eps(Complex *c) {
    return c->re * c->re + c->im * c->im + 1e-10;
}

__device__ float Complex_Arg(Complex *c) {
  return (c->per_fix + atan2f(c->im, c->re));
}

__device__ void Complex_Log(Complex *c) {
  Complex L_eps;
  Complex_Init(&L_eps, 0.5f * logf(Complex_Mag2eps(c)), Complex_Arg(c));
  Complex_Copy(c, &L_eps);
}

__device__ void Complex_Scale(Complex *c, float mul) {
    c->re = c->re * mul;
    c->im = c->im * mul;
}
  
__device__ void Complex_AtanH(Complex *c) {
    Complex D;
	Complex_Init(&D, c->re, c->im);
    Complex_Dec(&D);
    Complex_Neg(&D);
    Complex_Inc(c);
    Complex_Div(c, &D);
    Complex_Log(c);
    Complex_Scale(c, 0.5f);
}

__device__ void Complex_AcotH(Complex *c) {
   Complex_Recip(c);
   Complex_AtanH(c);
}

__device__ void Complex_Flip(Complex *c) {
    float r2 = c->im;
    float i2 = c->re;
    c->re = r2;
    c->im = i2;
  }
  
__device__ void Complex_Sqr(Complex *c) {
  float r2 = c->re * c->re - c->im * c->im;
  float i2 = 2.f * c->re * c->im;
  c->re = r2;
  c->im = i2;
}  

  
__device__ void Complex_Add(Complex *c, Complex *zz) {
  c->re += zz->re;
  c->im += zz->im;
}

__device__ void Complex_Sub(Complex *c, Complex *zz) {
  c->re -= zz->re;
  c->im -= zz->im;
}


__device__ void Complex_Mul(Complex *c, Complex *zz) {
   if (zz->im == 0.0) {
      Complex_Scale(c, zz->re);
      return;
   }
   float  r2 = c->re * zz->re - c->im * zz->im;
   float  i2 = c->re * zz->im + c->im * zz->re;
   c->re = r2;
   c->im = i2;
}
    
  
__device__ void Complex_One(Complex *c) {
  c->re = 1.0f;
  c->im = 0.0f;
}

__device__ void Complex_Conj(Complex *c) {
  c->im = -c->im;
}


__device__ float Complex_Radius(Complex *c) {
    return hypotf(c->re, c->im);
}

__device__ void Complex_Sqrt(Complex *c) {
  float Rad = Complex_Radius(c);
  float sb = (c->im < 0) ? -1.f : 1.f;
  c->im = sb * sqrtf(0.5f * (Rad - c->re));
  c->re = sqrtf(0.5f * (Rad + c->re));
  if (c->per_fix < 0)
    Complex_Neg(c);
}
  
  
__device__ void Complex_ToP(Complex *c, Complex *dst) {
  Complex_Init(dst, Complex_Radius(c), Complex_Arg(c));
}
  
  
__device__ void Complex_UnP(Complex *c, Complex *dst) {
  Complex_Init(dst, c->re * cosf(c->im), c->re * sinf(c->im));
}  
  
__device__ void Complex_Pow(Complex *c, float exp) {
    if (exp == 0.0f) {
      Complex_One(c);
      return;
    }
    float ex = fabsf(exp);
    if (exp < 0) {
      Complex_Recip(c);
    }
    if (ex == 0.5f) {
      Complex_Sqrt(c);
      return;
    }
    if (ex == 1.0f) {
      return;
    }
    if (ex == 2.0f) {
      Complex_Sqr(c);
      return;
    }
    // In general we need sin, cos etc
    Complex PF;
    Complex_ToP(c, &PF);
    PF.re = powf(PF.re, ex);
    PF.im = PF.im * ex;
	
	Complex PFU;	
	Complex_UnP(&PF, &PFU);	
    Complex_Copy(c, &PFU);
  }
  
 
__device__ void Complex_AsinH(Complex *c) {
  Complex D;
  Complex_Init(&D, c->re, c->im);
  Complex_Sqr(&D);
  Complex_Inc(&D);
  Complex_Pow(&D, 0.5f);
  Complex_Add(c, &D);
  Complex_Log(c);
}

__device__ void Complex_AsecH(Complex *c) {
   Complex_Recip(c);
   Complex_AsinH(c);
}

__device__ void Complex_Exp(Complex *c) {
   c->re = expf(c->re);
   Complex unp;
   Complex_UnP(c, &unp);
   Complex_Copy(c, &unp);
}

__device__ void Complex_AcosH(Complex *c) {
  Complex D;
  Complex_Init(&D, c->re, c->im);
  Complex_Sqr(&D);
  Complex_Dec(&D);
  Complex_Pow(&D, 0.5f);
  Complex_Add(c, &D);
  Complex_Log(c);
}

__device__ void Complex_AcosecH(Complex *c) {
   Complex_Recip(c);
   Complex_AcosH(c);
}

__device__ void Complex_SinH(Complex *c) {
    float rr = 0.0;
    float ri = 0.0;
    float er = 1.0;
    c->re = expf(c->re);
    er /= c->re;
    rr = 0.5 * (c->re - er);
    ri = rr + er;
    c->re = cosf(c->im) * rr;
    c->im = sinf(c->im) * ri;
}
  
__device__ void Complex_CosH(Complex *c) {
    float rr = 0.0;
    float ri = 0.0;
    float er = 1.0;
    c->re = expf(c->re);
    er /= c->re;
    rr = 0.5 * (c->re - er);
    ri = rr + er;
    c->re = cosf(c->im) * ri;
    c->im = sinf(c->im) * rr;
}

__device__ void Complex_Sin(Complex *c) {
    Complex_Flip(c);
    Complex_SinH(c);
    Complex_Flip(c);
}

__device__ void Complex_Cos(Complex *c) {
    Complex_Flip(c);
    Complex_CosH(c);
    Complex_Flip(c);
}

__device__ void Complex_Asin(Complex *c) {
    Complex_Flip(c);
    Complex_AsinH(c);
    Complex_Flip(c);
}

__device__ void Complex_Acos(Complex *c) {
    Complex_Flip(c);
    Complex_AsinH(c);
    Complex_Flip(c); 
    c->re = (M_PI_F/2.0) - (c->re);
    c->im = -(c->im); 
}

__device__ void Complex_Atan(Complex *c) { 
    Complex_Flip(c);
    Complex_AtanH(c);
    Complex_Flip(c);
} 


// Additional complex Functions

__device__ float Complex_arg (Complex z) {
    float result;
    result = atan2f(z.im, z.re);
    return result;
  }
  
__device__ float Complex_norm(Complex z) {
    double u = z.re;
    double v = z.im;
    return (u * u + v * v);
  }
  
__device__ float Complex_mag (Complex z) {
    return sqrtf(z.re*z.re + z.im*z.im);
 }
__device__ Complex Complex_plus (Complex a,Complex z) {
   Complex tmp;
   Complex_Init(&tmp, a.re+ z.re, a.im + z.im);
   return tmp;
  }
  
__device__ Complex Complex_minus (Complex a,Complex z) {
   Complex tmp;
   Complex_Init(&tmp, a.re - z.re, a.im - z.im);
   return tmp;
  }

__device__ Complex Complex_times (Complex a, float x) {
    Complex tmp;
    Complex_Init(&tmp,x*a.re,x*a.im);
	return tmp;
}

__device__ Complex Complex_times (Complex a, Complex z) {
   Complex tmp;
   Complex_Init(&tmp, a.re*z.re - a.im*z.im,a.re*z.im + a.im*z.re);
   return tmp;
}

__device__ Complex Complex_divideBy (Complex a, Complex z) {
    Complex tmp;
    float rz = Complex_mag(z);
    if(fabsf(rz) > 1.0e-12)
    {
	  Complex_Init(&tmp,(a.re * z.re + a.im * z.im)/(rz * rz),
                        (a.im * z.re - a.re * z.im)/(rz * rz));
    }	
	return tmp;
}

__device__ Complex Complex_sqrt(Complex z) {
    Complex tmp;
	float r = sqrtf(Complex_mag(z));
    float phi = Complex_arg(z)/2.0;
	Complex_Init(&tmp,r*cosf(phi),r*sinf(phi));
	return tmp;
}

__device__ Complex Complex_ln(Complex z) {
    Complex tmp;
    float rr = logf(Complex_mag(z))/logf(2.718);
    float ii = Complex_arg(z);
    Complex_Init(&tmp,rr,ii);
	return tmp;
}
  
__device__ Complex Complex_sin(Complex z) { 
    float r = sinf(z.re) * coshf(z.im);
    float i = cosf(z.re) * sinhf(z.im);
	Complex tmp;
	Complex_Init(&tmp,r,i);
    return tmp;
  }

__device__ Complex Complex_asinh(Complex zz)  {
    Complex i,z;
	Complex_Init(&i,1.0,0.0);
	z = Complex_plus(i,Complex_times(zz,zz));
    z = Complex_sqrt(z);
    z = Complex_plus(zz,z);
    z = Complex_ln(z);
    return z;
}

__device__ Complex Complex_asin(Complex z) {
    Complex j,zz;
	Complex_Init(&j,0.0, 1.0);
	Complex one;
	Complex_Init(&one,1.0,0.0);
	zz = Complex_minus(one , Complex_times(z,z));
    zz = Complex_sqrt(zz);
    zz = Complex_plus(zz,Complex_times(j,z));
    zz = Complex_times(Complex_times(j,Complex_ln(zz)), -1.0);
    return zz;
  }

__device__ Complex Complex_acos(Complex z) {
      Complex i,j,zz;
	  Complex_Init(&i,1.0,0.0);
	  Complex_Init(&j,0.0,1.0);
	  zz=Complex_minus(Complex_times(z,z),i);
      zz = Complex_sqrt(zz);
      zz = Complex_plus(z,zz);
      zz = Complex_times(Complex_times(j,Complex_ln(zz)),-1.0);
      return zz;
}

__device__ Complex Complex_tan(Complex z) {
    Complex tmp;
    float nenner = cosf(2.*z.re) + coshf(2*z.im);
    float r = sinf(2.*z.re) / nenner;
    float i = sinhf(2.*z.im) / nenner;
	Complex_Init(&tmp,r,i);
    return tmp;;
}


  
#endif

struct __align__(8) xForm
{
    float a;
    float b;
    float c;
    float d;
    float e;
    float f;
    float pa;
    float pb;
    float pc;
    float pd;
    float pe;
    float pf;
    float color;
    float symmetry;
    float weight;
    float opacity;
    float var_color;
    int   rotates;
#ifdef JWF_EXTENSIONS	
    float yzA;
    float yzB;
    float yzC;
    float yzD;
    float yzE;
    float yzF;
    float yzPa;
    float yzPb;
    float yzPc;
    float yzPd;
    float yzPe;
    float yzPf;
    float zxA;
    float zxB;
    float zxC;
    float zxD;
    float zxE;
    float zxF;
    float zxPa;
    float zxPb;
    float zxPc;
    float zxPd;
    float zxPe;
    float zxPf;
	int useXyz;
	int wfield_type;
	int wfield_input;
	float wfield_var_amount;
	int	wfield_param1_var_idx;
	int wfield_param1_param_idx;
	float wfield_param1_amount;
	int	wfield_param2_var_idx;
	int wfield_param2_param_idx;
	float wfield_param2_amount;
	int	wfield_param3_var_idx;
	int wfield_param3_param_idx;
	float wfield_param3_amount;
	float wfield_color_amount;
	float wfield_jitter_amount;
	int wfield_seed;
	int wfield_octaves;
	float wfield_gain;
	float wfield_lacunarity;
	float wfield_scale;
	int wfield_fractal_type;
    int wfield_cell_noise_dist_func;
    int wfield_cell_noise_ret_val;
#endif	
};

// each xform has a variable length list of active variations and each variation has its own specific variable sized varpar struct
struct __align__(16) VariationListNode
{                        // all of the lists are concatenated into a single buffer - a separate xformUsageIndex has the offset to the xform's first variation in this list
    uint variationID;    // the numeric value identifying the variation from the variation set - NOTE id of zero is used to signify end of list
    uint varparOffset;   // the offset in varpar union list for this variation's specific varpar struct
    uint enterGroup;     // the state transition that handles entering Pre, Normal, and Post variation groups
};

struct __align__(8) unAnimatedxForm
{
    float a;
    float b;
    float d;
    float e;
    int rotates;
};

struct __align__(16) FlameParams
{
    struct rgba background;
    float center[2];                //{x,y}
    float size[2];                    //size/(scale*zoom)
    float scale;
    float zoom;
    float cam_yaw;
    float cam_pitch;
    float cam_perspective;
    int   clipToNDC;
    float cam_dof;
    float cam_zpos;
    float cam_x;
    float cam_y;
    float cam_z;
    float cam_fov;
    float cam_near;
    float cam_orthowide;
    float hue;
    float numBatches;
    float quality;
    float desiredQuality;
    float rotation;
    float symmetryKind;
    float brightness;
    float gamma;
    float gammaThreshold;
    float alphaGamma;
    float vibrancy;
    unsigned int   numTrans;
    unsigned int   numFinal;
    int   useFinal;
    float supersampleWidth;
    int   frame;
    int   useXaos;
    int   oversample;
    float   highlightPower;
    int    estimatorRadius;			// default 7
    float  estimatorCurve;			// default 0.4
#ifdef JWF_EXTENSIONS
	float cam_roll;
	float cam_bank;
	float cam_xfocus;
	float cam_yfocus;
	float cam_zfocus;
	float cam_xpos;
	float cam_ypos;
	float cam_dist;
	float cam_dof_exponent;
	float cam_dof_area;
	bool legacy_dof;
	int dof_type;
	float dof_scale;
	float dof_fade;
    float balanceRed, balanceGreen, balanceBlue;
	float intensityAdjust;
	int render_pass;
	float z_buffer_scale, z_buffer_shift, z_buffer_bias;
#endif
};


struct Flame //  : public std::enable_shared_from_this<Flame>
{
    struct FlameParams params;
    int                numColors;
    struct xForm      *trans;
    struct xForm      *finals;
    struct rgba       *colorIndex;
    float             *colorLocations;
    float             *switchMatrix;
    //  std::vector<SharedVariationChain> xformVarChains;
    //  std::vector<SharedVariationChain> finalVarChains;
//#ifdef __cplusplus
//    Flame();
//    Flame(int numTrans,int paletteSize, int numFinal);
//    Flame(int numTrans,int paletteSize, int numFinal, int alignment);
//    Flame(const Flame &other);
//    Flame(const Flame &other, int alignment);
//    
//    void Clone(Flame** target);
//    void CloneAligned(Flame** target, int alignment);
//    void deleteChildren();
//    void prepareSwitchMatrix (float *brick);
//    ~Flame();
//#endif
} __attribute__ ((aligned (16)));
#endif

__VARPAR_STRUCT_DECLS__


#define PI 3.141592653589793f

#ifndef FLAM4_KERNAL_CUH
#define FLAM4_KERNAL_CUH

struct __align__(16) point
{
    float x;
    float y;
    float z;
    float pal;
#ifdef JWF_EXTENSIONS
	float colorR, colorG, colorB, colorA;
	bool useRgb;
	bool doHide;
#endif
};
#endif

#ifndef FOR_2D
struct  __align__(16) CameraViewProperties
{
    float matrix[16];
    float yaw;
    float pitch;
    float roll;
    float perspective;
    float dof;
    float zpos;
    float cosRoll;
    float sinRoll;
    float camWidth;
    float camHeight;
    float centerX;
    float centerY;
    int   clipToNDC;
    float rotatedViewOffsetx;
    float rotatedViewOffsety;
#ifdef JWF_EXTENSIONS
	float bank;
	float focusX;
	float focusY;
	float focusZ;
	float camPosX;
	float camPosY;
	float camPosZ;
	float camDist;
	float camDOFExponent;
	float camDOFArea;
	float camDOF;
	bool legacyDOF;
	int dofType;
	float dofScale;
	float dofFade;	
#endif	
};
#endif

// so it can be precompiled as part of the build for syntax checking
#ifndef KERNEL_RUNTIME
#define WARP_SIZE 32
#define NUM_POINTS 64
#endif
#define ADD_EPSILON +epsilon
//#define ADD_EPSILON +1.e-7f
//#define ADD_EPSILON +1.e-10f
//#define ADD_EPSILON

#define WARPS_PER_BLOCK 2
#define BLOCKDIM WARPS_PER_BLOCK*WARP_SIZE

#define RANDFLOAT() randFloat(randStates)
#define RANDINT()   randInt(randStates)

__VARIATION_INDEX_DEFINES__

__constant__ FlameParams d_g_Flame;
__constant__ xForm d_g_Xforms[MAX_XFORMS+MAX_XFORMS];  // has both normal & final xforms

__constant__ unsigned int shift1[4] = {6, 2, 13, 3};
__constant__ unsigned int shift2[4] = {13, 27, 21, 12};
__constant__ unsigned int shift3[4] = {18, 2, 7, 13};
__constant__ unsigned int offset[4] = {4294967294, 4294967288, 4294967280, 4294967168};

texture<uchar4, hipTextureType1D, hipReadModeNormalizedFloat> texRef;

__VARIATION_FUNCTION_PROTOTYPES__


__device__ float sqrf(float x) {
  return x*x;
} 

__device__ unsigned int TausStep(unsigned int z, int S1, int S2, int S3, unsigned int M)
{
    unsigned int b = (((z << S1) ^ z) >> S2);
    return (((z &M) << S3) ^ b);
}

__device__ unsigned int randInt(unsigned int *randStates)
{
    unsigned int index = threadIdx.x;
    randStates[index&(WARP_SIZE-1)] = TausStep(randStates[index&(WARP_SIZE-1)], shift1[index&3], shift2[index&3],shift3[index&3],offset[index&3]);
    return (randStates[(index)&(WARP_SIZE-1)]^randStates[(index+1)&(WARP_SIZE-1)]^randStates[(index+2)&(WARP_SIZE-1)]^randStates[(index+3)&(WARP_SIZE-1)]);
}

__device__ float randFloat(unsigned int *randStates)
//This function returns a random float in [0,1] and updates seed
{
    unsigned int y = randInt(randStates);
    return __int_as_float((y&0x007FFFFF)|0x3F800000)-1.0f;
}

__device__ float randFloatWarp(unsigned int *randStates, uint index)
//This function is a workaround for getting a warp wide rand number
{
    randInt(randStates);
    return __int_as_float((randStates[index]&0x007FFFFF)|0x3F800000)-1.0f;
}

__device__ int linear_range_search(float* X, float x, int n)
{
    int index = 256;
    for (int j = n-1; j >= 0; j--) {
        index = (x <= X[j+1]) ? j : index;
    }
    return index;
}

__device__ float curveAdjust(float x,
        float* X,
        float* A,
        float* B,
        float* C,
        float* D,
        uint cpCount)
{
    int index    = linear_range_search(X, x, cpCount - 1);
    float result = (x <= X[0]) ? A[0] : A[cpCount - 1];

    if (index >= 0 && index < (int)cpCount - 1) {
        float t = x - X[index];
        result = A[index] + B[index]*t + C[index]*t*t + D[index]*t*t*t;
    }
    return result;
}


__device__ float4 RGBtoHSV(float4 color)
{
    float r = color.x;
    float g = color.y;
    float b = color.z;
    float mx = fmaxf(fmaxf(r,g),b);
    float mn = fminf(fminf(r,g),b);
    float h,s,v;
    if (mx == mn)
        h = 0.0f;
    else if (mx == r)
        h = .16666666667f*(g-b)/(mx-mn);
    else if (mx == g)
        h = .16666666667f*(b-r)/(mx-mn)+.33333333f;
    else
        h = .16666666667f*(r-g)/(mx-mn)+.66666667f;
    h = h-floorf(h);
    if (mx == 0.0f)
        s = 0.0f;
    else
        s = (mx-mn)/(mx);
    v = mx;
    if (v > 1.0f) // clamp to 1.f if to high value
        v = 1.0f;
    return make_float4(h,s,v,color.w);
}

__device__ float4 RGBtoHSVHueAdjusted(float4 color)
{
    float r = color.x;
    float g = color.y;
    float b = color.z;
    float mx = fmaxf(fmaxf(r,g),b);
    float mn = fminf(fminf(r,g),b);
    float h,s,v;
    if (mx == mn)
        h = 0.0f;
    else if (mx == r)
        h = .16666666667f*(g-b)/(mx-mn);
    else if (mx == g)
        h = .16666666667f*(b-r)/(mx-mn)+.33333333f;
    else
        h = .16666666667f*(r-g)/(mx-mn)+.66666667f;
    h = h-floorf(h);
    if (mx == 0.0f)
        s = 0.0f;
    else
        s = (mx-mn)/(mx);
    v = mx;
    if (v > 1.0f)
    {
        if (h < .33333333f)
        {
            h += (.16666667f-h)*(1.0f-powf(.75f,v-1.0f));
        }
        else if (h < 0.5f)
        {
            h += (h-0.5f)*(1.0f-powf(.75f,v-1.0f));
        }
        else if (h > 0.8333333f)
        {
            h += (h-0.8333333f)*(1.0f-powf(.75f,v-1.0f));
        }
        //float l = .2126f*r+.7152f*g+.0722f*b;
        //float l = (40.0f*r+20.0f*g+b)/61.0f;
        float l = 0.4f+0.4f*cosf(2.0f*PI*(h-0.16666666667f));
        s = fminf(s*powf(1.0f/v,0.6f*(1.0f-l)),s);
    }
    return make_float4(h,s,v,color.w);
}

__device__ float4 HSVtoRGB(float4 color)
{
    float h = color.x;
    float s = color.y;
    float v = color.z;
    float r,g,b;
    int hi = ((int)floorf(h*6.0f))%6;
    float f = h*6.0f-floorf(h*6.0f);
    float p = v*(1.0f-s);
    float q = v*(1.0f-f*s);
    float t = v*(1.0f-(1.0f-f)*s);
    switch (hi)
    {
        case 0:
        {
            r = v;
            g = t;
            b = p;
        }break;
        case 1:
        {
            r = q;
            g = v;
            b = p;
        }break;
        case 2:
        {
            r = p;
            g = v;
            b = t;
        }break;
        case 3:
        {
            r = p;
            g = q;
            b = v;
        }break;
        case 4:
        {
            r = t;
            g = p;
            b = v;
        }break;
        case 5:
        {
            r = v;
            g = p;
            b = q;
        }break;
    }
    return make_float4(r,g,b,color.w);
}

__device__ float4 read_imageStepMode(float4 * image, int length, float index)
{
    float clampedIndex = index - floorf(index);
    float scaledIndex = clampedIndex*(float)(length - 1);
    int iLow = floorf(scaledIndex);
    return image[iLow];
}

__device__ float sinhcoshf(float theta, float* ch)
{
    float expt = expf(theta);
    float exptinv = 1.0f / expt;
    *ch =  (expt + exptinv) * 0.5f;
    return (expt - exptinv) * 0.5f;
}

__VARIATION_FUNCTIONS__

#ifdef ADD_FEATURE_WFIELDS
__device__ float calcWFieldIntensity(float3 *__wFieldPos, struct xForm* xform) {
        switch(xform->wfield_type) {
#ifdef ADD_FEATURE_CELLULAR_NOISE
             case 1: // Cellular Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = Cellular;
                  switch(xform->wfield_cell_noise_ret_val) {
                    case 0: // Cell value
                      noise.m_cellularReturnType = CellValue;
                      break;
                    case 1: // Distance1
                      noise.m_cellularReturnType = Distance;
                      break;
                    case 2: // Distance2
                      noise.m_cellularReturnType = Distance2;
                      break;
                    case 3: // Dist add
                      noise.m_cellularReturnType = Distance2Add;
                      break;
                    case 4: // Dist sub
                      noise.m_cellularReturnType = Distance2Sub;
                      break;
                    case 5: // Dist mul
                      noise.m_cellularReturnType = Distance2Mul;
                      break;
                    case 6: // Dist div
                      noise.m_cellularReturnType = Distance2Div;
                      break;
                    default:
                      noise.m_cellularReturnType = Distance2;
                      break;
                  }
                  noise.m_cellularDistanceFunction = xform->wfield_cell_noise_dist_func == 1 ? Manhattan : xform->wfield_cell_noise_dist_func == 2 ? Natural : Euclidean;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_CUBIC_NOISE
             case 2: // Cubic Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = Cubic;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_CUBIC_NOISE
             case 3: // Cubic Fractal Noise
                {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = CubicFractal;
                  noise.m_octaves = xform->wfield_octaves;
                  noise.m_lacunarity = xform->wfield_lacunarity;
                  noise.m_gain = xform->wfield_gain;
                  noise.m_fractalType = xform->wfield_fractal_type == 1 ? Billow : xform->wfield_fractal_type == 2 ? RigidMulti : FBM;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_PERLIN_NOISE
             case 4: // Perlin Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = Perlin;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_PERLIN_NOISE
             case 5: // Perlin Fractal Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = PerlinFractal;
                  noise.m_octaves = xform->wfield_octaves;
                  noise.m_lacunarity = xform->wfield_lacunarity;
                  noise.m_gain = xform->wfield_gain;
                  noise.m_fractalType = xform->wfield_fractal_type == 1 ? Billow : xform->wfield_fractal_type == 2 ? RigidMulti : FBM;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_SIMPLEX_NOISE
             case 6: // Simplex Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = Simplex;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_SIMPLEX_NOISE
             case 7: // Simplex Fractal Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = SimplexFractal;
                  noise.m_octaves = xform->wfield_octaves;
                  noise.m_lacunarity = xform->wfield_lacunarity;
                  noise.m_gain = xform->wfield_gain;
                  noise.m_fractalType = xform->wfield_fractal_type == 1 ? Billow : xform->wfield_fractal_type == 2 ? RigidMulti : FBM;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_VALUE_NOISE
             case 8: // Value Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = Value;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_VALUE_NOISE
             case 9: // Value Fractal Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_frequency = xform->wfield_scale;
                  noise.m_noiseType = ValueFractal;
                  noise.m_octaves = xform->wfield_octaves;
                  noise.m_lacunarity = xform->wfield_lacunarity;
                  noise.m_gain = xform->wfield_gain;
                  noise.m_fractalType = xform->wfield_fractal_type == 1 ? Billow : xform->wfield_fractal_type == 2 ? RigidMulti : FBM;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
#ifdef ADD_FEATURE_WHITE_NOISE
             case 10: // White Noise
               {
                  FastNoise noise;
                  fastNoise_init(&noise);
                  noise.m_seed = xform->wfield_seed;
                  noise.m_noiseType = WhiteNoise;
                  fastNoise_prepare(&noise);
                  return getNoise(&noise, __wFieldPos->x, __wFieldPos->y, __wFieldPos->z);
               }
               break;
#endif
/*
             case 11: // Image Map -> not supported
*/
             default:
               return  0.f;
               break;
           }
  //return  0.f;
}

#endif


__device__ void iteratePoint(struct VariationListNode *varUsageList,
                float *varpars,
                struct xForm* xform,
                uint xformIndex,
                float epsilon,
                struct point *fromPoint,
                struct point *activePoint,
                unsigned int *randStates,
                uint *permutations,
#ifdef JWF_EXTENSIONS
                float4* palette,
                uint numColors,
#endif
                float4 *gradients)
{
    unsigned int index = threadIdx.x;
    activePoint[index] = *fromPoint;

    float s0 = xform->symmetry;
    float s1 = .5f-.5f*s0;
    float __pal = (activePoint[index].pal+xform->color)*s1+activePoint[index].pal*s0;
    float pal0 = __pal;

     float __x, __y, __z;
	 float __colorR=0.f, __colorG=0.f, __colorB=0.f, __colorA=0.f;
	 bool __doHide = false;
	 bool __useRgb = false;
	
	 if(xform->useXyz) {
		 __x = xform->a*activePoint[index].x+xform->b*activePoint[index].y;
		 __y = xform->d*activePoint[index].x+xform->e*activePoint[index].y;
		 __z = activePoint[index].z;

         float nx, ny, nz;  
         ny = xform->yzA * __y + xform->yzB * __z;
         nz = xform->yzD * __y + xform->yzE * __z;
         __y = ny;
         __z = nz;

         nx = xform->zxA * __x + xform->zxB * __z;
         nz = xform->zxD * __x + xform->zxE * __z;
         __x = nx;
         __z = nz;
		 
		 __x += xform->c + xform->zxC;
		 __y += xform->f + xform->yzC;
		 __z += xform->yzF + xform->zxF;
	 }
	 else {
		 __x = xform->a*activePoint[index].x+xform->b*activePoint[index].y+xform->c;
		 __y = xform->d*activePoint[index].x+xform->e*activePoint[index].y+xform->f;
		 __z = activePoint[index].z; // 3d hack does not transform them here
	 }
////WFIELD
     bool __useWFields = false;
     float __wFieldValue;
     float __wFieldAmountScale;
#ifdef ADD_FEATURE_WFIELDS
     if(xform->wfield_type>0) {
         float3 __wFieldPos;
         if(xform->wfield_input == 1) { // Position
           __wFieldPos = make_float3(activePoint[index].x, activePoint[index].y, activePoint[index].z);
         }
         else {
           __wFieldPos = make_float3(__x, __y, __z); // Affine
         }
         __wFieldValue = calcWFieldIntensity(&__wFieldPos, xform);
          if(fabs(__wFieldValue)>EPSILON) {
                __useWFields = true;
                __wFieldAmountScale = 1.0f + __wFieldValue * xform->wfield_var_amount;
          }
          else {
                __wFieldAmountScale = 1.f;
          }
     }
     else {
         __wFieldAmountScale = 1.f;
         __wFieldValue = 0.f;
     }
#else
     __wFieldAmountScale = 1.f;
     __wFieldValue = 0.f;
#endif


////WFIELD
	float __r2, __r, __rinv, __phi, __theta;
    float __px = __x;  // note that enterGroup action will handle resetting these to zero -- also works correctly for xforms with NO variations set
    float __py = __y;
    float __pz = __z;

	float __x0 = __x;
	float __y0 = __y;
	float __z0 = __z;

	bool __was_pre = 0;


			__r2 = __x * __x + __y * __y;
			__r = sqrtf(__r2);
			__rinv = 1.0f / __r;

			__phi = atan2f(__x, __y);
			__theta = .5f*PI - __phi;
			if (__theta > PI)
				__theta -= 2.0f*PI;


     __px = 0.f;
     __py = 0.f;
     __pz = 0.f;


    uint varIndex = 0;
    uint varCounter = 0;
    while ((varIndex = varUsageList->variationID) != 0) {
        float *varparCluster = &varpars[varUsageList->varparOffset];
        switch (varIndex) {
            //Now apply the Variations
            __VARIATION_SWITCH_CASES__
            default:
            break;
        }
        varUsageList++;
        varCounter++;
    }

    if(xform->useXyz) {
      float px = xform->pa*__px+xform->pb*__py;
      float py = xform->pd*__px+xform->pe*__py;
      float pz = __pz;
	  
	  float nx, ny, nz;
	    
      ny = xform->yzPa * py + xform->yzPb * pz;
      nz = xform->yzPd * py + xform->yzPe * pz;
      py = ny;
      pz = nz;
	 
	  nx = xform->zxPa * px + xform->zxPb * pz;
      nz = xform->zxPd * px + xform->zxPe * pz;
      px = nx;
      pz = nz;
	  
      activePoint[index].x = px + xform->pc + xform->zxPc;
      activePoint[index].y = py + xform->pf + xform->yzPc;
      activePoint[index].z = pz + xform->yzPf + xform->zxPf;
	 
	}
	else {
      activePoint[index].x = xform->pa*__px+xform->pb*__py+xform->pc;
      activePoint[index].y = xform->pd*__px+xform->pe*__py+xform->pf;
      activePoint[index].z = __pz;
    }
    //    activePoint[index].z=z;  // 3d hack does not transform them here
#ifdef ADD_FEATURE_WFIELDS_JITTER
    if(__useWFields && fabsf(xform->wfield_jitter_amount) > EPSILON) {
      float jitterIntensity = 0.1 * xform->wfield_jitter_amount;
      {
          float3 __wFieldJitterPos = make_float3(activePoint[index].x, activePoint[index].y, activePoint[index].z);
          float __wFieldJitterValue = calcWFieldIntensity(&__wFieldJitterPos, xform);
          activePoint[index].x += (__wFieldJitterValue * jitterIntensity);
      }
      {
          float3 __wFieldJitterPos = make_float3(activePoint[index].y, activePoint[index].x, activePoint[index].z);
          float __wFieldJitterValue = calcWFieldIntensity(&__wFieldJitterPos, xform);
          activePoint[index].y += (__wFieldJitterValue * jitterIntensity);
      }
      {
          float3 __wFieldJitterPos = make_float3(activePoint[index].z, activePoint[index].x, activePoint[index].y);
          float __wFieldJitterValue = calcWFieldIntensity(&__wFieldJitterPos, xform);
          activePoint[index].z += (__wFieldJitterValue * jitterIntensity);
      }
    }
#endif
    if (d_g_Flame.symmetryKind != 0.0f && xformIndex < d_g_Flame.numTrans) // does not apply to final xform
    {
        if (d_g_Flame.symmetryKind > 0.0f)
        {
            float rn;
            rn = randFloat(randStates);
            float sina, cosa;
            sincosf(2.0f*PI*floorf(rn*d_g_Flame.symmetryKind)/d_g_Flame.symmetryKind, &sina, &cosa);

            __x = cosa*activePoint[index].x-sina*activePoint[index].y;
            __y = sina*activePoint[index].x+cosa*activePoint[index].y;
            activePoint[index].x = __x;
            activePoint[index].y = __y;
        }
        else
        {
            //pick a random symmetry plane and reflect across it.
            float rn;
            float rn2;
            rn2 = randFloat(randStates);
            rn = randFloat(randStates);
            float sina, cosa;
            sincosf(2.0f*PI*floorf(rn*d_g_Flame.symmetryKind)/d_g_Flame.symmetryKind, &sina, &cosa);

            __x = cosa*activePoint[index].x-sina*activePoint[index].y;
            __y = sina*activePoint[index].x+cosa*activePoint[index].y;
            if (rn2>0.5f)
                __x = -__x;
            activePoint[index].x = __x;
            activePoint[index].y = __y;
        }
    }
    activePoint[index].pal =  pal0 + xform->var_color * (__pal - pal0);
#ifdef JWF_EXTENSIONS	
    activePoint[index].doHide = __doHide;
	activePoint[index].useRgb = __useRgb;
    if(__useRgb) {
	  activePoint[index].colorR = __colorR; 
	  activePoint[index].colorG = __colorG; 
	  activePoint[index].colorB = __colorB; 
	  activePoint[index].colorA = __colorA; 
	}
#ifdef ADD_FEATURE_WFIELDS
	////WFIELD
	if(__useWFields && fabsf(xform->wfield_color_amount)>EPSILON) {
		  if(activePoint[index].pal<0.f)
		    activePoint[index].pal = 0.f;
		  else if(activePoint[index].pal>1.0f)
            activePoint[index].pal = 1.f;
          activePoint[index].pal *= (1.0f + __wFieldValue *  xform->wfield_color_amount * 0.1);
		  if(activePoint[index].pal<0.f)
		    activePoint[index].pal = 0.f;
		  else if(activePoint[index].pal>1.0f) 
            activePoint[index].pal = 1.f;  		  
        }
#endif
	////WFIELD

#endif
}

#ifndef FOR_2D
__device__ void applyRotation(struct point* point, float rotatedViewOffsetx, float rotatedViewOffsety)
{
    point->x += rotatedViewOffsetx;
    point->y += rotatedViewOffsety;
}

__device__ void applyOnlyCamera(struct point* point, float srcX, float srcY, float srcZ, float zdist, float zr)
{
    point->x = srcX / zr;
    point->y = srcY / zr;
}

__device__ void applyDOFAndCamera(struct point* point, float srcX, float srcY, float srcZ, float zdist, float zr, int dofType, float dofScale, float dofFade, float camDOF_10, unsigned int *randStates)
{    
    float fade;    
	if (dofFade <= 1.e-6f) {
      fade = 1.0f;
    }
    else if (dofFade >= 1.0f - 1.e-6f) {
      fade = randFloat(randStates);
    }
    else {
      fade = randFloat(randStates) <= dofFade ? randFloat(randStates) : 1.0f;
    }

	float dr = fade * camDOF_10 * zdist * dofScale;
#ifdef ADD_FEATURE_DOF
    switch(dofType) {
	  case 0: // BUBBLE
	  default:
	    {
			float a = 2 * PI * randFloat(randStates);
			float dsina, dcosa;
			sincosf(a, &dsina, &dcosa);
			point->x = (srcX + dr * dcosa) / zr;
			point->y = (srcY + dr * dsina) / zr;
			break;
		}
	   case 1: // SINEBLUR
         {
		   float power = 4.2f;
		   float a = 2 * PI * randFloat(randStates);
		   float dsina, dcosa;
		   sincosf(a, &dsina, &dcosa);

           dr *= (acosf(expf(logf(randFloat(randStates)) * power) * 2.0f - 1.0f) / PI);

   		   point->x = (srcX + dr * dcosa) / zr;
		   point->y = (srcY + dr * dsina) / zr;
		   break;
         }		 
	}
#endif
}



__device__ void projectJWF(struct point *p, struct CameraViewProperties *properties, unsigned int *randStates)
{
#ifndef JWF_EXTENSIONS	
    float px, py, pz, pw;
    px = properties->matrix[0]*p->x + properties->matrix[4]*p->y + properties->matrix[8]*p->z+ properties->matrix[12];
    py = properties->matrix[1]*p->x + properties->matrix[5]*p->y + properties->matrix[9]*p->z+ properties->matrix[13];
    pz = properties->matrix[2]*p->x + properties->matrix[6]*p->y + properties->matrix[10]*p->z+ properties->matrix[14];
    pw = properties->matrix[3]*p->x + properties->matrix[7]*p->y + properties->matrix[11]*p->z+ properties->matrix[15];

    // handle Apophysis perspective perspective == 0.f ==> Ortho, perspective == 1.f ==> Normal Perspective
    pw  = 1.f + (pw - 1.f) * properties->perspective;

    if (properties->dof > 1.e-6f) {
        float zdist = properties->zpos - pz;
        float t     = randFloat(randStates) * 2.f * M_PI_F;
        float dr    = randFloat(randStates) * 0.1f * properties->dof * zdist;
        float sina, cosa;
        sincosf(t, &sina, &cosa);

        if (zdist > 0.f) {
            p->x = (px + dr*cosa)/pw;
            p->y = (py + dr*sina)/pw;
            p->z = pz/pw;
        }
        else {
            p->x = px/pw;
            p->y = py/pw;
            p->z = pz/pw;
        }
    }
    else {
        p->x = px/pw;
        p->y = py/pw;
        p->z = pz/pw;
    }
#else 
    float camPointX = properties->matrix[0]*p->x + properties->matrix[4]*p->y + properties->matrix[8]*p->z;
    float camPointY = properties->matrix[1]*p->x + properties->matrix[5]*p->y + properties->matrix[9]*p->z;
    float camPointZ = properties->matrix[2]*p->x + properties->matrix[6]*p->y + properties->matrix[10]*p->z;
	  float camDOF_10 = 0.1 * properties->camDOF; 	  
	  float area = properties->camDOFArea;
      float fade = properties->camDOFArea / 2.25f;
      float areaMinusFade = area - fade;
  	  
      camPointX += properties->camPosX;
      camPointY += properties->camPosY;
      camPointZ += properties->camPosZ;

      float zr = 1.0f - properties->perspective * camPointZ + properties->camPosZ;
      if (zr < 1.e-6f) {
        zr = 1.e-6f;
      }
      p->z = camPointZ;
#ifdef ADD_FEATURE_DOF
      if (properties->camDOF > 1.e-6f) {
        if (properties->legacyDOF) {
          float zdist = properties->camDist - camPointZ;
          if (zdist > 0.0f) {
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, randStates);
          }
          else {
            applyOnlyCamera(p, camPointX, camPointY, camPointZ, zdist, zr);
          }
        }
        else {
          float xdist = (camPointX - properties->focusX);
          float ydist = (camPointY - properties->focusY);
          float zdist = (camPointZ - properties->focusZ);

          float dist = powf(xdist * xdist + ydist * ydist + zdist * zdist, 1.0f / properties->camDOFExponent );
          if (dist > area) {
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, randStates);
          }
          else if (dist > areaMinusFade) {
            double scl = smootherstep(0.0f, 1.0f, (dist - areaMinusFade) / fade);
            double sclDist = scl * dist;
            applyDOFAndCamera(p, camPointX, camPointY, camPointZ, zdist, zr, properties->dofType, properties->dofScale, properties->dofFade, camDOF_10, randStates);
          }
          else {
            applyOnlyCamera(p, camPointX, camPointY, camPointZ, zdist, zr);
          }
        }
      }
      else {
#endif // ADD_FEATURE_DOF
        p->x = camPointX / zr;
        p->y = camPointY / zr;
#ifdef ADD_FEATURE_DOF
      }
#endif // ADD_FEATURE_DOF
#endif	
}

#else
__device__ 
    void applyRotation(struct point* Point, float cosRotation, float sinRotation)
{
    float x,y;
    x = Point->x-d_g_Flame.center[0];
    y = Point->y-d_g_Flame.center[1];

    Point->x = x*cosRotation - y*sinRotation + d_g_Flame.center[0];
    Point->y = x*sinRotation + y*cosRotation + d_g_Flame.center[1];
}

#endif

__device__
float4 read_image(float4 * image, int length, float index)
{
    float clampedIndex = index - floor(index);
    float scaledIndex = clampedIndex*(float)(length - 1);
    int iLow = floor(scaledIndex);
    int iHigh = ceil(scaledIndex);
    float iFract = scaledIndex - floor(scaledIndex);
    float4 c0 = image[iLow];
    float4 c1 = image[iHigh];
    return make_float4(iFract*c1.x+(1.0f-iFract)*c0.x, 
                        iFract*c1.y+(1.0f-iFract)*c0.y, 
                        iFract*c1.z+(1.0f-iFract)*c0.z, 
                        iFract*c1.w+(1.0f-iFract)*c0.w);
}

extern "C" __global__
void reductionKernal(unsigned* buffer,
                     unsigned length,
                     unsigned * result)
{
    volatile __shared__ unsigned scratch[BLOCKDIM];

    unsigned global_index = blockIdx.x*blockDim.x+threadIdx.x;

    // Perform parallel reduction
    int local_index      = threadIdx.x;
    scratch[local_index] = global_index < length ? buffer[global_index] : 0;
	__syncthreads();

    for(int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (local_index < offset) {
            scratch[local_index] += scratch[local_index + offset];
        }
    	__syncthreads();
    }
    if (local_index == 0) {
        result[blockIdx.x] = scratch[0];
    }
}

extern "C" __global__ void iteratePointsKernal(struct VariationListNode *d_g_varUsages,
                                uint *d_g_varUsageIndexes,
                                float *varpars,
                                float *d_g_switchMatrix,
#ifndef FOR_2D
                                struct CameraViewProperties *d_g_Camera,
#endif
                                float4* renderTarget,
                                struct point* points,
                                uint* pointIterations,
                                uint* perThreadRandSeeds,
                                float4* palette,
                                uint numColors,
                                int  paletteStepMode,
                                float epsilon,
                                uint fuseIterations,
                                int xDim,
                                int yDim,
                                uint *startingXform,
                                uint *markedCounts,
                                uint *pixelCounts,
                                uint xformPointPoolSize,
                                uint *permutations,
                                float4 *gradients,
                                uint *shuffle,
                                uint *iterationCount)
{
    __shared__ struct point activePoint[BLOCKDIM];
    __shared__ uint         randStates[WARP_SIZE];
#ifdef PARALLEL_SELECT
    __shared__ uint         rw[2];
#endif
    uint maxR            = d_g_Flame.numTrans - 1;
    uint index           = threadIdx.x; // blockDim.x should be 2 * WARP_SIZE
    uint blockIndex      = blockIdx.y * gridDim.x + blockIdx.x;
    const int ix         = (blockDim.x * blockIndex) + index;
    const uint warpIndex = ix/WARP_SIZE;
    //    const uint whichWarp = warpIndex % WARPS_PER_BLOCK;
    const uint whichWarp = warpIndex & 1;
#ifdef FOR_2D
    float sinRotation, cosRotation;
    sincosf(d_g_Flame.rotation, &sinRotation, &cosRotation);
#endif

    // want to measure the actual number of batches, suspect driver is not executing all batches
    if (blockIndex == 0 && index == 0)
        iterationCount[0] += 1;

    // Iterate some points!
    randStates[index&(WARP_SIZE-1)] = perThreadRandSeeds[ix];

    // want randStates buffers to be populated for entire block before continuing
	__syncthreads();

    uint fromXform = startingXform[warpIndex];
    uint toXform   = 0;
	
	
    for (int j = 0; j < NUM_ITERATIONS; j++)
    {
        //Pick xform for this iteration
#ifdef PARALLEL_SELECT
        float w       = randFloatWarp(randStates, whichWarp);
        rw[whichWarp] = 0;
        __syncthreads();

        uint offset = 0;
        uint windex = index & (WARP_SIZE-1);
        while (windex + offset <= maxR) {
            uint lixw = fromXform * d_g_Flame.numTrans + windex + offset;
            if (windex + offset > 0 && w >= d_g_switchMatrix[lixw - 1] && w < d_g_switchMatrix[lixw]) {
                rw[whichWarp] = windex + offset;
            }
            offset += WARP_SIZE;
        }
        __syncthreads();
        uint r = rw[whichWarp];
        toXform = r;
#else
        float w;
        w=randFloatWarp(randStates, whichWarp);
        uint r    = 0;
        while ((r < maxR) && (w > d_g_switchMatrix[fromXform * d_g_Flame.numTrans + r]))
        {
            r++;
        }
        toXform = r;
#endif
        //Now each thread chooses a point at random from the point pool.  This is done to allow each point to have a seperate xform path while retaining SIMD
        uint p               = shuffle[index + NUM_POINTS * j];
        uint fromPointIndex  = fromXform * xformPointPoolSize + NUM_POINTS*blockIndex + p;
        uint toPointIndex    = toXform   * xformPointPoolSize + NUM_POINTS*blockIndex + p;
        uint iterations      = pointIterations[fromPointIndex];
        uint varUsagesIndex  = d_g_varUsageIndexes[r];

        struct point fromPoint = points[fromPointIndex];

        //Iterate the chosen point and store it back to the pool
        iteratePoint(&d_g_varUsages[varUsagesIndex],
                     varpars,
                     &d_g_Xforms[r],
                     r,
                     epsilon,
                     &fromPoint,
                     activePoint,
                     randStates,
                     permutations,
#ifdef JWF_EXTENSIONS
                     palette,
                     numColors,
#endif
                     gradients);

#ifndef FOR_2D
        if (! isfinite(activePoint[index].x + activePoint[index].y + activePoint[index].z)) {
            // test to add back a random point (ala Flam3) to get Flam3 like images in borderline cases
            activePoint[index].x = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].y = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].z = 0; //2.f*randFloat(randStates) - 1.f;
            iterations           = 0;
        }

#else
        if (! isfinite(activePoint[index].x + activePoint[index].y)) {
            // test to add back a random point (ala Flam3) to get Flam3 like images in borderline cases
            activePoint[index].x = 2.f*randFloat(randStates) - 1.f;
            activePoint[index].y = 2.f*randFloat(randStates) - 1.f;
            iterations           = 0;
        }

#endif
        ++iterations;
        struct point toPoint = activePoint[index]; // capture point state before final xform application
	

        if (iterations >= fuseIterations) { // dont store until fuse for each point is finished
            markedCounts[ix]++;  // keep track of number of iterations that could mark (versus unmarked because of unfused points)

            //Prepare the point for displey.  First the final transformation is applied
            if (d_g_Flame.useFinal) {
                for (uint fIndex = 0; fIndex < d_g_Flame.numFinal; fIndex++) {
                    uint varUsagesIndex  = d_g_varUsageIndexes[d_g_Flame.numTrans + fIndex];
                    struct point tempPoint = activePoint[index];
					float pal_save = activePoint[index].pal;
                    iteratePoint(&d_g_varUsages[varUsagesIndex],
                                 varpars,
                                 &d_g_Xforms[d_g_Flame.numTrans + fIndex],
                                 d_g_Flame.numTrans  + fIndex,
                                 epsilon,
                                 &tempPoint,
                                 activePoint,
                                 randStates,
                                 permutations,
#ifdef JWF_EXTENSIONS
                                 palette,
                                 numColors,
#endif
                                 gradients);
								activePoint[index].pal = pal_save; 
                }
            }

#ifdef JWF_EXTENSIONS
         if(!activePoint[index].doHide) {
#endif			


#ifndef FOR_2D
            projectJWF(&activePoint[index], d_g_Camera, randStates);
			if(fabsf(d_g_Camera->pitch)<1.0e-6 && fabsf(d_g_Camera->yaw)<1.0e-6 && fabsf(d_g_Camera->bank)<1.0e-6) {
              applyRotation(&activePoint[index], d_g_Camera->rotatedViewOffsetx, d_g_Camera->rotatedViewOffsety);
			}
#else
            applyRotation( &activePoint[index], cosRotation, sinRotation);
#endif
            //Finally, we randomly jitter the point within a 1/2 pixel radius to obtain antialiasing
            float dr;
            dr = randFloat(randStates);
            dr = expf(d_g_Flame.supersampleWidth*sqrtf(-logf(dr)))-1.0f;
            float rn;
            rn = randFloat(randStates);
            float dtheta = (rn)*2.0f*PI;

            // mark the histogram
            float z = (d_g_Flame.clipToNDC != 0) ? activePoint[index].z : 0.f;
            int x,y;
            float sina, cosa;
            sincosf(dtheta, &sina, &cosa);

            x = floorf((((activePoint[index].x-d_g_Flame.center[0])/d_g_Flame.size[0]+.5f)*(float)xDim)+dr*cosa);
            y = floorf(((-(activePoint[index].y-d_g_Flame.center[1])/d_g_Flame.size[1]+.5f)*(float)yDim)+dr*sina);
            //And render the point to the accumulation buffer
            if ((z >= -1.f) && (z <= 1.f) && (x < xDim)&&(y < yDim)&&(x>=0)&&(y>=0)) {
              if(d_g_Flame.render_pass==1) {
                 float zBias = d_g_Flame.z_buffer_bias;
                 float zScale = d_g_Flame.z_buffer_scale;
                 float zOffset = d_g_Flame.z_buffer_shift;

                  // negative zScale: white to black (black near camera, white background)
                  int grayValue;
                  if (zScale < 0.0) {
                      int lvl = (int)(-zScale * (activePoint[index].z + zOffset) * 32767.0 + 32767.5);
                      if (lvl < 0) {
                        lvl = 0;
                      } else if (lvl > 0xffff) {
                        lvl = 0xffff;
                      }
                      grayValue = 0xffff - lvl & 0xffff;
                  }
                  // positive zScale: black to white (white near camera, black background, which is the default)
                  else {
                      int lvl = (int)(zScale * (activePoint[index].z + zOffset) * 32767.0 + 32767.5);
                      if (lvl < 0) {
                        lvl = 0;
                      } else if (lvl > 0xffff) {
                        lvl = 0xffff;
                      }
                      grayValue = lvl & 0xffff;
                  }

                  float zValue = (float)grayValue / 65536.0f;

                 if(zValue > renderTarget[y*xDim+x].x) {
                   renderTarget[y*xDim+x].x = zValue;
                   renderTarget[y*xDim+x].y = 0.0f;
                   renderTarget[y*xDim+x].z = 0.0f;
                   renderTarget[y*xDim+x].w = 1.0f;
                   pixelCounts[y*xDim+x]++;
                 }
              }
              else {
                  float4 output;
    #ifdef JWF_EXTENSIONS
                if(activePoint[index].useRgb) {
                   output.x = activePoint[index].colorR;
                   output.y = activePoint[index].colorG;
                   output.z = activePoint[index].colorB;
                   output.w = activePoint[index].colorA;
                 }
                 else {
    #endif
                    if (paletteStepMode)
                        output = read_imageStepMode(palette, numColors, activePoint[index].pal);
                    else
                        output = read_image(palette, numColors, activePoint[index].pal);
                    // output = tex1D(texRef,activePoint[threadIdx.x].pal);

    #ifdef JWF_EXTENSIONS
                } // if(activePoint[index].useRgb) {
                if(d_g_Flame.intensityAdjust!=1) {
                  output.x *= d_g_Flame.intensityAdjust;
                  output.y *= d_g_Flame.intensityAdjust;
                  output.z *= d_g_Flame.intensityAdjust;
                  output.w *= d_g_Flame.intensityAdjust;
                }
    #endif

    #ifdef USE_ATOMICS
                    float *ptr = (float *)&(renderTarget[y*xDim+x]);
                    atomicAdd(ptr,     output.x*d_g_Xforms[r].opacity);
                    atomicAdd(ptr + 1, output.y*d_g_Xforms[r].opacity);
                    atomicAdd(ptr + 2, output.z*d_g_Xforms[r].opacity);
                    atomicAdd(ptr + 3, output.w*d_g_Xforms[r].opacity);
                    atomicAdd(&pixelCounts[y*xDim+x], 1);
    #else
                    renderTarget[y*xDim+x].x += output.x*d_g_Xforms[r].opacity;
                    renderTarget[y*xDim+x].y += output.y*d_g_Xforms[r].opacity;
                    renderTarget[y*xDim+x].z += output.z*d_g_Xforms[r].opacity;
                    renderTarget[y*xDim+x].w += output.w*d_g_Xforms[r].opacity;
                    pixelCounts[y*xDim+x]++;
    #endif
                }
#ifdef JWF_EXTENSIONS
             }
         } // if(activePoint[index].doHide==0) {
#endif			
			
        }
        pointIterations[toPointIndex] = iterations;
        points[toPointIndex]          = toPoint;
        fromXform                     = toXform;
        // essential to prevent cross-warp corruption with linear
    	__syncthreads();
        
    }
    startingXform[warpIndex] = toXform;
    perThreadRandSeeds[ix]   = randStates[index&(WARP_SIZE-1)];
}

extern "C" __global__ void postProcessStep1Kernal(
                                float4* renderTarget,
                                float4* accumBuffer,
                                uint xDim,
                                uint yDim,
                                int blocksY,
                                float fuseCompensation)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;

    if ((ix < xDim)&&(iy < yDim)) {
        if(d_g_Flame.render_pass==1) {
            float4 rgba = accumBuffer[iy*xDim+ix];
            rgba.y = rgba.x;
            rgba.z = rgba.x;
            rgba.w = 1.0;
            accumBuffer[iy*xDim+ix] = rgba;
        }
        else {
            float k1 = (d_g_Flame.brightness*268.0f)/255.0f;
            float area = fabsf(d_g_Flame.size[0]*d_g_Flame.size[1]);
            float k2 = ((float)(xDim*yDim))/(area*fuseCompensation*((float)(NUM_ITERATIONS))*d_g_Flame.numBatches*32.f*1024.0f*((float)blocksY/32.f));
            float4 rgba = accumBuffer[iy*xDim+ix];
            float a = (k1* logf(1.0f+k2*rgba.w));
            float ls = a/rgba.w;
            rgba.x = ls*rgba.x;
            rgba.y = ls*rgba.y;
            rgba.z = ls*rgba.z;

            accumBuffer[iy*xDim+ix] = rgba;
        }
    }
}

extern "C" __global__ void postProcessStep2Kernal(
                                float4* renderTarget,
                                float4* accumBuffer,
                                uint xDim,
                                uint yDim,
                                int blocksY,
                                float fuseCompensation,
                                float4 adjust)
{

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim)) {
        if(d_g_Flame.render_pass==1) {
            float4 rgba = accumBuffer[iy*xDim+ix];
                renderTarget[iy*xDim+ix].x=rgba.x;
                renderTarget[iy*xDim+ix].y=rgba.y;
                renderTarget[iy*xDim+ix].z=rgba.z;
                renderTarget[iy*xDim+ix].w=1.0;
        }
        else {
            float k1   = (d_g_Flame.brightness*268.0f)/255.0f;
            float area = fabsf(d_g_Flame.size[0]*d_g_Flame.size[1]);
            float _k2  = ((float)(xDim*yDim))/
                    (area*fuseCompensation*((float)(NUM_ITERATIONS))*d_g_Flame.numBatches*32.f*1024.0f*((float)blocksY/32.f));
            float gammaThreshold = d_g_Flame.gammaThreshold;
            float gamma          = d_g_Flame.gamma;
            float alphaGamma     = d_g_Flame.alphaGamma;

            float4 k2   = make_float4(_k2/adjust.x, _k2/adjust.y, _k2/adjust.z, _k2/adjust.w);
            float4 rgba = accumBuffer[iy*xDim+ix];

            float4 a = make_float4(k1 * logf(1.0f + k2.x*rgba.w),
                                   k1 * logf(1.0f + k2.y*rgba.w),
                                   k1 * logf(1.0f + k2.z*rgba.w),
                                   k1 * logf(1.0f + k2.w*rgba.w));


            float4 fraction = make_float4(a.x/gammaThreshold,
                                          a.y/gammaThreshold,
                                          a.z/gammaThreshold,
                                          a.w/gammaThreshold);

            float4 alpha = make_float4(powf(a.x, 1.0f/gamma-1.0f),
                                        powf(a.y, 1.0f/gamma-1.0f),
                                        powf(a.z, 1.0f/gamma-1.0f),
                                        powf(a.w, 1.0f/gamma-1.0f));


            float alphaTx =  (1.f - fraction.x) * a.x * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.x * alpha.x;
            float alphaTy =  (1.f - fraction.y) * a.y * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.y * alpha.y;
            float alphaTz =  (1.f - fraction.z) * a.z * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.z * alpha.z;
            float alphaTw =  (1.f - fraction.w) * a.w * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.w * alpha.w;

            float4 alphaT   = make_float4(alphaTx, alphaTy, alphaTz, alphaTw);

            float4 ls = make_float4(
                d_g_Flame.vibrancy * (a.x < gammaThreshold ? alphaT.x : alpha.x),
                d_g_Flame.vibrancy * (a.y < gammaThreshold ? alphaT.y : alpha.y),
                d_g_Flame.vibrancy * (a.z < gammaThreshold ? alphaT.z : alpha.z),
                d_g_Flame.vibrancy * (a.w < gammaThreshold ? alphaT.w : alpha.w));

            float4 sign = make_float4(
                rgba.x >= 0.f ? 1.f : -1.f,
                rgba.y >= 0.f ? 1.f : -1.f,
                rgba.z >= 0.f ? 1.f : -1.f,
                rgba.w >= 0.f ? 1.f : -1.f);

            rgba.x        = ls.x*rgba.x + (1.0f-d_g_Flame.vibrancy)*sign.x*powf(fabsf(rgba.x), 1.0f/gamma);
            rgba.y        = ls.y*rgba.y + (1.0f-d_g_Flame.vibrancy)*sign.y*powf(fabsf(rgba.y), 1.0f/gamma);
            rgba.z        = ls.z*rgba.z + (1.0f-d_g_Flame.vibrancy)*sign.z*powf(fabsf(rgba.z), 1.0f/gamma);
            rgba.w        = ls.w*rgba.w + (1.0f-d_g_Flame.vibrancy)*sign.w*powf(fabsf(rgba.w), 1.0f/gamma);

            alpha.x       = powf(a.x, 1.0f/gamma);
            alpha.y       = powf(a.y, 1.0f/gamma);
            alpha.z       = powf(a.z, 1.0f/gamma);
            alpha.w       = powf(a.w, 1.0f/gamma);

            alphaT.x      = (1.f - fraction.x) * a.x * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.x * alpha.x;
            alphaT.y      = (1.f - fraction.y) * a.y * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.y * alpha.y;
            alphaT.z      = (1.f - fraction.z) * a.z * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.z * alpha.z;
            alphaT.w      = (1.f - fraction.w) * a.w * (powf(gammaThreshold, gamma)/gammaThreshold) + fraction.w * alpha.w;

            alpha.x      = fminf((a.x < gammaThreshold ? alphaT.x : alpha.x), 1.f);
            alpha.y      = fminf((a.y < gammaThreshold ? alphaT.y : alpha.y), 1.f);
            alpha.z      = fminf((a.z < gammaThreshold ? alphaT.z : alpha.z), 1.f);
            alpha.w      = fminf((a.w < gammaThreshold ? alphaT.w : alpha.w), 1.f);


            float alphaCw  = powf(a.w, 1.0f/alphaGamma);
            float alphaTCw =  (1.f - fraction.w) * a.w * (powf(gammaThreshold, alphaGamma)/gammaThreshold) + fraction.w * alphaCw;
            alphaCw        = fminf((a.w < gammaThreshold ? alphaTCw : alphaCw), 1.f);


            if (d_g_Flame.highlightPower >= 0.f) {
                rgba = RGBtoHSVHueAdjusted(rgba);
                if (rgba.z > 1.0f)
                {
                    //rgba.y /= rgba.z;
                    rgba.z = 1.0f;
                }
                rgba = HSVtoRGB(rgba);
            }
            if (isfinite(rgba.x + rgba.y + rgba.z + rgba.w))
            {
                renderTarget[iy*xDim+ix].x=rgba.x+renderTarget[iy*xDim+ix].x*(1.0f-alpha.x);
                renderTarget[iy*xDim+ix].y=rgba.y+renderTarget[iy*xDim+ix].y*(1.0f-alpha.y);
                renderTarget[iy*xDim+ix].z=rgba.z+renderTarget[iy*xDim+ix].z*(1.0f-alpha.z);
                renderTarget[iy*xDim+ix].w=alphaCw;
            }
            else {
                renderTarget[iy*xDim+ix].w=0.f;
            }
        }
    }
}

extern "C" __global__ void colorCurveRGB3ChannelsKernal(float4* accumBuffer,
                                        float* X,
                                        float* A,
                                        float* B,
                                        float* C,
                                        float* D,
                                        uint xDim,
                                        uint yDim,
                                        uint cpCount)

{
    __shared__ float xs[256];
    __shared__ float as[256];
    __shared__ float bs[256];
    __shared__ float cs[256];
    __shared__ float ds[256];

    cpCount    = cpCount > 256 ? 256 : cpCount;
    uint index = threadIdx.x;

    for (uint i = 0; index+i < cpCount; i += blockDim.x) {
        xs[index+i] = X[index+i];
        as[index+i] = A[index+i];
        cs[index+i] = C[index+i];
    }
    for (uint i = 0; index+i < cpCount-1; i += blockDim.x) {
        bs[index+i] = B[index+i];
        ds[index+i] = D[index+i];
    }
	__syncthreads();

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float4 rgba = accumBuffer[iy*xDim+ix];
        // sRGB luma   0.212656 R  0.715158 G  0.072186 B
        // AdobeRGB luma
        float preluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        preluma       /= rgba.w;
        float postluma = curveAdjust(preluma, xs, as, bs, cs, ds, cpCount);

        if (preluma != 0.f) {
            rgba.x = postluma/preluma * rgba.x;
            rgba.y = postluma/preluma * rgba.y;
            rgba.z = postluma/preluma * rgba.z;
            rgba.w = postluma/preluma * rgba.w;
        }
        else {
            rgba.x = postluma;
            rgba.y = postluma;
            rgba.z = postluma;
            rgba.w = postluma;
        }
        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void colorCurveRGBChannelKernal(float4* accumBuffer,
                                        float* X,
                                        float* A,
                                        float* B,
                                        float* C,
                                        float* D,
                                        uint xDim,
                                        uint yDim,
                                        uint cpCount,
                                        uint channel)
{
    __shared__ float xs[256];
    __shared__ float as[256];
    __shared__ float bs[256];
    __shared__ float cs[256];
    __shared__ float ds[256];

    cpCount    = cpCount > 256 ? 256 : cpCount;
    uint index = threadIdx.x;

    for (uint i = 0; index+i < cpCount; i += blockDim.x) {
        xs[index+i] = X[index+i];
        as[index+i] = A[index+i];
        cs[index+i] = C[index+i];
    }
    for (uint i = 0; index+i < cpCount-1; i += blockDim.x) {
        bs[index+i] = B[index+i];
        ds[index+i] = D[index+i];
    }
	__syncthreads();

    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        float4 rgba = accumBuffer[iy*xDim+ix];

        float preluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        switch (channel) {
            default:
            case 0:
            rgba.x = rgba.w * curveAdjust(rgba.x/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 1:
            rgba.y = rgba.w * curveAdjust(rgba.y/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 2:
            rgba.z = rgba.w * curveAdjust(rgba.z/rgba.w, xs, as, bs, cs, ds, cpCount);
            break;
            case 3:
            break;
        }
        float postluma  = 0.297361f * rgba.x + 0.627355f * rgba.y + 0.075285f * rgba.z;
        // maintain same luminance afterwards
        if (preluma != 0)
            rgba.w *= postluma/preluma;
        else
            rgba.w  = postluma;

        accumBuffer[iy*xDim+ix] = rgba;
    }
}

extern "C" __global__ void setBufferKernal(float4* renderTarget, float4 value, uint xDim, uint yDim)
{
    //This kernal simply fills the render target with value
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;

    if ((ix < xDim)&&(iy < yDim))
        renderTarget[iy*xDim+ix] = value;
}

extern "C" __global__ void FlexibleDensityEstimationKernal(float* output, float* input, unsigned int xDim, unsigned int yDim,
                                                            float baseThreshold, int radius)
{
    __shared__ float filterLocal[4*(DENSITY_KERNAL_RADIUS*2+16)*(DENSITY_KERNAL_RADIUS*2+16)];
    unsigned int index    = threadIdx.x;
    const unsigned int ix = blockDim.x*blockIdx.x + index;
    const unsigned int iy = blockDim.y*blockIdx.y + threadIdx.y;
    const unsigned int lidx = (blockDim.x+radius*2)*threadIdx.y + index;
    //First, we load up the block of pixels we will be working on into shared memory
    for (unsigned int y = 0; y < blockDim.y+radius*2-threadIdx.y; y += blockDim.y)
    {
        for(unsigned int x = 0; x < blockDim.x+radius*2-index; x += blockDim.x)
        {
            unsigned toIndex4   = 4*(lidx+x+y*(blockDim.x+radius*2));
            //clamped addressing
            unsigned fromIndex4 = 4*(max(min(ix+x-radius,xDim-1),0U)+max(min(iy+y-radius,yDim-1),0U)*xDim);
            filterLocal[toIndex4]     = input[fromIndex4];
            filterLocal[toIndex4 + 1] = input[fromIndex4 + 1];
            filterLocal[toIndex4 + 2] = input[fromIndex4 + 2];
            filterLocal[toIndex4 + 3] = input[fromIndex4 + 3];
        }
    }
    __syncthreads();
    //Next, apply the actual filter
    if ((ix < xDim)&& (iy < yDim))
    {
        unsigned idx = 4*(ix+iy*xDim);
        float pntw = input[idx + 3];
        float sumx = 0.f;
        float sumy = 0.f;
        float sumz = 0.f;
        float sumw = 0.f;

        float count = 0.0f;
        for (int y = 0; y < radius*2+1; y++)
        {
            for (int x = 0; x<radius*2+1; x++)
            {
                int cellIdx   = 4*(lidx+x+y*(blockDim.x+2*radius));
                float invDist = 1.0f/(((float)(x-radius)*(float)(x-radius)+(float)(y-radius)*(float)(y-radius))+1.0f);
                float deviation = fabsf(erff((filterLocal[cellIdx + 3]-pntw)/(sqrtf(8.0f*pntw)+5.0f)));
                if (deviation<=powf(baseThreshold*.9f,sqrtf(1.0f/invDist))*powf(pntw+1.0f,-0.25f))
                {
                    sumx += filterLocal[cellIdx]*invDist;
                    sumy += filterLocal[cellIdx + 1]*invDist;
                    sumz += filterLocal[cellIdx + 2]*invDist;
                    sumw += filterLocal[cellIdx + 3]*invDist;
                    count += invDist;
                }
            }
        }
        sumx/=count;
        sumy/=count;
        sumz/=count;
        sumw/=count;
        //And store the result
        output[idx]     = isfinite(sumx)  ? sumx : 0.f;
        output[idx + 1] =  isfinite(sumy) ? sumy : 0.f;
        output[idx + 2] =  isfinite(sumz) ? sumz : 0.f;
        output[idx + 3] =  isfinite(sumw) ? sumw : 0.f;
    }
}

extern "C" __global__ void RGBA128FtoRGBA32UKernal(uchar4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    //This kernal converts a 32bit per channel floating point image to a 8bit per channel integer image
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_uchar4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*255.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*255.0f);
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*255.0f;
                output[iy*xDim+ix] = make_uchar4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    alpha);
            }
            else
                output[iy*xDim+ix]=make_uchar4(0,0,0,0);
            }
        else
        {
        if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
            output[iy*xDim+ix] = make_uchar4(
                fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*255.0f,
                fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*255.0f,
                fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*255.0f,
                255);
        else
            output[iy*xDim+ix]=make_uchar4(0,0,0,255);
        }
    }
}

extern "C" __global__ void RGBA128FtoBGRA32UKernal(uchar4* output, float4* input, uint xDim, uint yDim)
{
    //This kernal converts a 32bit per channel floating point image to a 8bit per channel integer image
    // in BGRA format for little endian Intel with premultiplied alpha
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
        output[iy*xDim+ix] = make_uchar4(
            fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*255.0f, // blue
            fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*255.0f, // green
            fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*255.0f, // red
            255);
        else
            output[iy*xDim+ix]=make_uchar4(0,0,0,255);
    }
}

extern "C" __global__ void RGBA128FtoRGBA64UKernal(ushort4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    //This kernal converts a 32bit per channel floating point image to a 16bit per channel integer image
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_ushort4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f,
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f);
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f)*65535.0f;
                output[iy*xDim+ix] = make_ushort4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    alpha);
            }
            else
                output[iy*xDim+ix]=make_ushort4(0,0,0,0);
        }
        else
        {
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
                output[iy*xDim+ix] = make_ushort4(
                    fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f)*65535.0f,
                    fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f)*65535.0f,
                    fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f)*65535.0f,
                    65535);
            else
                output[iy*xDim+ix]=make_ushort4(0,0,0,65535);
        }
    }
}

extern "C" __global__ void RGBA128FtoRGBA128FKernal(float4* output, float4* input, uint xDim, uint yDim, int useAlpha)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        if (useAlpha)
        {
//            // straight alpha
//            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f))
//                output[iy*xDim+ix] = make_float4(
//                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f),
//                    fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f));
            // premultiplied alpha
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w) && (input[iy*xDim+ix].w!=0.0f)) {
                float alpha = fmaxf(fminf(input[iy*xDim+ix].w,1.0f),0.0f);
                output[iy*xDim+ix] = make_float4(
                    fmaxf(fminf(input[iy*xDim+ix].x/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].y/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                    fmaxf(fminf(input[iy*xDim+ix].z/input[iy*xDim+ix].w,1.0f),0.0f)*alpha,
                alpha);
            }
            else
                output[iy*xDim+ix]=make_float4(0.f,0.f,0.f,0.f);
        }
        else
        {
            if (isfinite(input[iy*xDim+ix].x+input[iy*xDim+ix].y+input[iy*xDim+ix].z+input[iy*xDim+ix].w))
                output[iy*xDim+ix] = make_float4(
                    fmaxf(fminf(input[iy*xDim+ix].x,1.0f),0.0f),
                    fmaxf(fminf(input[iy*xDim+ix].y,1.0f),0.0f),
                    fmaxf(fminf(input[iy*xDim+ix].z,1.0f),0.0f),
                    1.f);
            else
                output[iy*xDim+ix]=make_float4(0.f,0.f,0.f,1.f);
        }
    }
}

extern "C" __global__ void MergeKernal(float4* accum, float4* input, uint xDim, uint yDim)
{
    const uint ix = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        accum[iy*xDim+ix].x += input[iy*xDim+ix].x;
        accum[iy*xDim+ix].y += input[iy*xDim+ix].y;
        accum[iy*xDim+ix].z += input[iy*xDim+ix].z;
        accum[iy*xDim+ix].w += input[iy*xDim+ix].w;
    }
}

extern "C" __global__ void readChannelKernel(float* output, float4* input, uint xDim, uint yDim, uint channel)
{
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        switch (channel) {
            default:
            case 0:
            output[iy*xDim+ix] = input[iy*xDim+ix].x;
            break;
            case 1:
            output[iy*xDim+ix] = input[iy*xDim+ix].y;
            break;
            case 2:
            output[iy*xDim+ix] = input[iy*xDim+ix].z;
            break;
            case 3:
            output[iy*xDim+ix] = input[iy*xDim+ix].w;
            break;
        }
    }
}

extern "C" __global__ void writeChannelKernel(float4* output, float* input, uint xDim, uint yDim, uint channel)
{
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim)&&(iy < yDim))
    {
        switch (channel) {
            default:
            case 0:
            output[iy*xDim+ix].x = input[iy*xDim+ix];
            break;
            case 1:
            output[iy*xDim+ix].y = input[iy*xDim+ix];
            break;
            case 2:
            output[iy*xDim+ix].z = input[iy*xDim+ix];
            break;
            case 3:
            output[iy*xDim+ix].w = input[iy*xDim+ix];
            break;
        }
    }
}

extern "C" __global__ void writeChannelStripedKernel(float4* output,
                                        float* input,
                                        uint xDim,
                                        uint yDim,
                                        uint channel,
                                        uint supersample)
{
    const uint resampledXdim = xDim / supersample;
    const uint ix   = (blockDim.x*blockIdx.x)+threadIdx.x;
    const uint iy   = (blockDim.y*blockIdx.y)+threadIdx.y;
    if ((ix < xDim) && (iy < yDim) && (ix % supersample == 0) && (iy % supersample == 0))
    {
        const uint x = ix / supersample;
        const uint y = iy / supersample;
        switch (channel) {
            default:
            case 0:
            output[y*resampledXdim+x].x = input[iy*xDim+ix];
            break;
            case 1:
            output[y*resampledXdim+x].y = input[iy*xDim+ix];
            break;
            case 2:
            output[y*resampledXdim+x].z = input[iy*xDim+ix];
            break;
            case 3:
            output[y*resampledXdim+x].w = input[iy*xDim+ix];
            break;
        }
    }
}


///////////////////////////////////////////////////////////////////////////////
/*
* Copyright 1993-2007 NVIDIA Corporation.  All rights reserved.
* OpenCL port & resampling kernels Copyright 2014 Centcom Inc. All rights reserved.
*
* NOTICE TO USER:
*
* This source code is subject to NVIDIA ownership rights under U.S. and
* international Copyright laws.  Users and possessors of this source code
* are hereby granted a nonexclusive, royalty-free license to use this code
* in individual and commercial software.
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL,
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS
* OF USE, DATA OR PROFITS,  WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE
* OR OTHER TORTIOUS ACTION,  ARISING OUT OF OR IN CONNECTION WITH THE USE
* OR PERFORMANCE OF THIS SOURCE CODE.
*
* U.S. Government End Users.   This source code is a "commercial item" as
* that term is defined at  48 C.F.R. 2.101 (OCT 1995), consisting  of
* "commercial computer  software"  and "commercial computer software
* documentation" as such terms are  used in 48 C.F.R. 12.212 (SEPT 1995)
* and is provided to the U.S. Government only as a commercial end item.
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the
* source code with only those rights set forth herein.
*
* Any use of this source code in individual and commercial software must
* include, in the user documentation and internal comments to the code,
* the above Disclaimer and U.S. Government End Users Notice.
*/

////////////////////////////////////////////////////////////////////////////////
// Row convolution filter
////////////////////////////////////////////////////////////////////////////////

// Assuming ROW_TILE_W, KERNEL_RADIUS_ALIGNED and dataW
// are multiples of coalescing granularity size,
// all global memory operations are coalesced in convolutionRowGPU()
#define            ROW_TILE_W 128
#define KERNEL_RADIUS_ALIGNED 16

// Assuming COLUMN_TILE_W and dataW are multiples
// of coalescing granularity size, all global memory operations
// are coalesced in convolutionColumnGPU()
#define COLUMN_TILE_W 16
#define COLUMN_TILE_H 48

extern "C" __global__ void convolutionRowsKernel(float *d_Kernel,
                                float *d_Result,
                                float *d_Data,
                                int dataW,
                                int dataH,
                                int KERNEL_RADIUS
){
    //Data cache
    __shared__ float data[KERNEL_RADIUS_ALIGNED + ROW_TILE_W + KERNEL_RADIUS_ALIGNED];

    //Current tile and apron limits, relative to row start
    const int         tileStart = blockIdx.x * ROW_TILE_W;
    const int           tileEnd = tileStart + ROW_TILE_W - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataW - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataW - 1);

    //Row start index in d_Data[]
    const int          rowStart = blockIdx.y * dataW;

    //Aligned apron start. Assuming dataW and ROW_TILE_W are multiples
    //of half-warp size, rowStart + apronStartAligned is also a
    //multiple of half-warp size, thus having proper alignment
    //for coalesced d_Data[] read.
    const int apronStartAligned = tileStart - KERNEL_RADIUS_ALIGNED;

    const int loadPos = apronStartAligned + threadIdx.x;
    //Set the entire data cache contents
    //Load global memory values, if indices are within the image borders,
    //or initialize with zeroes otherwise
    if(loadPos >= apronStart){
        const int smemPos = loadPos - apronStart;

        // out of bounds set to 0
        // data[smemPos] =
        //     ((loadPos >= apronStartClamped) && (loadPos <= apronEndClamped)) ? d_Data[rowStart + loadPos] : 0;

        // reflected at boundary
        // data[smemPos] =
        // loadPos < apronStartClamped ? d_Data[rowStart + apronStartClamped + apronStartClamped - loadPos] :
        //  (loadPos > apronEndClamped ? d_Data[rowStart + apronEndClamped   + apronEndClamped   - loadPos] :
        //     d_Data[rowStart + loadPos]);

        // clamp to border
        data[smemPos] =
            loadPos < apronStartClamped ? d_Data[rowStart + apronStartClamped] :
            (loadPos > apronEndClamped ? d_Data[rowStart + apronEndClamped] :
            d_Data[rowStart + loadPos]);
    }


    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
	__syncthreads();

    const int writePos = tileStart + threadIdx.x;

    //Assuming dataW and ROW_TILE_W are multiples of half-warp size,
    //rowStart + tileStart is also a multiple of half-warp size,
    //thus having proper alignment for coalesced d_Result[] write.
    if(writePos <= tileEndClamped){
        const int smemPos = writePos - apronStart;
        float sum = 0;

        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += data[smemPos + k] * d_Kernel[KERNEL_RADIUS - k];

        d_Result[rowStart + writePos] = isfinite(sum) ? sum : 0.f;
    }
}

extern "C" __global__ void convolutionColumnsKernel(float *d_Kernel,
                                        float *d_Result,
                                        float *d_Data,
                                        int dataW,
                                        int dataH,
                                        int smemStride,
                                        int gmemStride,
                                        int KERNEL_RADIUS
){
    //Data cache
    __shared__ float data[COLUMN_TILE_W * (KERNEL_RADIUS_ALIGNED + COLUMN_TILE_H + KERNEL_RADIUS_ALIGNED)];

    //Current tile and apron limits, in rows
    const int         tileStart = blockIdx.y * COLUMN_TILE_H;
    const int           tileEnd = tileStart + COLUMN_TILE_H - 1;
    const int        apronStart = tileStart - KERNEL_RADIUS;
    const int          apronEnd = tileEnd   + KERNEL_RADIUS;

    //Clamp tile and apron limits by image borders
    const int    tileEndClamped = min(tileEnd, dataH - 1);
    const int apronStartClamped = max(apronStart, 0);
    const int   apronEndClamped = min(apronEnd, dataH - 1);

    //Current column index
    const int       columnStart = blockIdx.x * COLUMN_TILE_W + threadIdx.x;

    //Shared and global memory indices for current column
    int smemPos    = threadIdx.y * COLUMN_TILE_W + threadIdx.x;
    int gmemPos    = (apronStart + threadIdx.y) * dataW + columnStart;
    int gmemPosMin = columnStart;
    int gmemPosMax = columnStart + dataW * (dataH - 1);

    //Cycle through the entire data cache
    //Load global memory values, if indices are within the image borders,
    //or initialize with zero otherwise
    for(int y = apronStart + threadIdx.y; y <= apronEnd; y += blockDim.y){
        // out of bounds set to 0
        //data[smemPos] =
        //    ((y >= apronStartClamped) && (y <= apronEndClamped)) ? d_Data[gmemPos] : 0;

        // reflected at boundary
        // data[smemPos] =
        //     y < apronStartClamped ? d_Data[gmemPosMin + (apronStartClamped - y) * dataW] :
        //    (y > apronEndClamped   ? d_Data[gmemPosMax + (apronEndClamped   - y) * dataW] : d_Data[gmemPos]);

        // clamp to border
        data[smemPos] =
            y < apronStartClamped ? d_Data[gmemPosMin] :
            (y > apronEndClamped   ? d_Data[gmemPosMax] : d_Data[gmemPos]);

        smemPos += smemStride;
        gmemPos += gmemStride;
    }

    //Ensure the completness of the loading stage
    //because results, emitted by each thread depend on the data,
    //loaded by another threads
	__syncthreads();

    //Shared and global memory indices for current column
    smemPos = (threadIdx.y + KERNEL_RADIUS) * COLUMN_TILE_W + threadIdx.x;
    gmemPos = (tileStart + threadIdx.y) * dataW + columnStart;

    //Cycle through the tile body, clamped by image borders
    //Calculate and output the results
    for(int y = tileStart + threadIdx.y; y <= tileEndClamped; y += blockDim.y){
        float sum = 0;

        for(int k = -KERNEL_RADIUS; k <= KERNEL_RADIUS; k++)
            sum += data[smemPos + k * COLUMN_TILE_W] * d_Kernel[KERNEL_RADIUS - k];

        d_Result[gmemPos] = isfinite(sum) ? sum : 0.f;
        smemPos += smemStride;
        gmemPos += gmemStride;
    }
}
